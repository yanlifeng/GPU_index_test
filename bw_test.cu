#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

int main() {
    const size_t size = 1024ll * 1024 * 1024;  // 100 MB
    void *host_ptr, *device_ptr;

    // Allocate pageable host memory (default)
    //host_ptr = malloc(size);
    hipHostAlloc(&host_ptr, size, hipHostMallocDefault);
    memset(host_ptr, 0, size);
    hipMalloc(&device_ptr, size);

    // Warm up
    hipMemcpy(device_ptr, host_ptr, size, hipMemcpyHostToDevice);

    // Measure H2D bandwidth
    auto start = std::chrono::high_resolution_clock::now();
    hipMemcpy(device_ptr, host_ptr, size, hipMemcpyHostToDevice);
    auto end = std::chrono::high_resolution_clock::now();
    double h2d_time = std::chrono::duration<double>(end - start).count();
    std::cout << "Host-to-Device Bandwidth: " << (size / 1e9) / h2d_time << " GB/s" << std::endl;

    // Measure D2H bandwidth
    start = std::chrono::high_resolution_clock::now();
    hipMemcpy(host_ptr, device_ptr, size, hipMemcpyDeviceToHost);
    end = std::chrono::high_resolution_clock::now();
    double d2h_time = std::chrono::duration<double>(end - start).count();
    std::cout << "Device-to-Host Bandwidth: " << (size / 1e9) / d2h_time << " GB/s" << std::endl;

    // Cleanup
    hipFree(device_ptr);
    hipFree(host_ptr);

    return 0;
}

