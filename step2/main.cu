#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <stdexcept>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstring> // For strerror
#include <sys/time.h>
#include <thread>
#include <omp.h>
#include <unistd.h>
#include "kseq++/kseq++.hpp"

#include "FastxStream.h"
#include "FastxChunk.h"
#include "DataQueue.h"
#include "Formater.h"

#include "index.hpp"
#include "indexparameters.hpp"
#include "cmdline.hpp"
#include "exceptions.hpp"
#include "io.hpp"
#include "randstrobes.hpp"
#include "refs.hpp"
#include "logger.hpp"
#include "pc.hpp"
#include "readlen.hpp"
#include "my_struct.hpp"

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#define my_bucket_index_t StrobemerIndex::bucket_index_t

#define rescue_threshold 100

inline double GetTime() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_sec + (double) tv.tv_usec / 1000000;
}

InputBuffer get_input_buffer(const CommandLineOptions &opt) {
    if (opt.is_SE) {
        return InputBuffer(opt.reads_filename1, "", opt.chunk_size, false);

    } else if (opt.is_interleaved) {
        if (opt.reads_filename2 != "") {
            throw BadParameter("Cannot specify both --interleaved and specify two read files");

        }
        return InputBuffer(opt.reads_filename1, "", opt.chunk_size, true);

    } else {
        return InputBuffer(opt.reads_filename1, opt.reads_filename2, opt.chunk_size, false);

    }

}

int producer_pe_fastq_task(std::string file, std::string file2, rabbit::fq::FastqDataPool &fastqPool,
                           rabbit::core::TDataQueue<rabbit::fq::FastqDataPairChunk> &dq) {
    rabbit::fq::FastqFileReader *fqFileReader;
    fqFileReader = new rabbit::fq::FastqFileReader(file, fastqPool, false, file2);
    int n_chunks = 0;
    int line_sum = 0;
    while (true) {
        rabbit::fq::FastqDataPairChunk *fqdatachunk = new rabbit::fq::FastqDataPairChunk;
        fqdatachunk = fqFileReader->readNextPairChunk();
        if (fqdatachunk == NULL) break;
        //std::cout << "readed chunk: " << n_chunks << std::endl;
        dq.Push(n_chunks, fqdatachunk);
        n_chunks++;
    }

    dq.SetCompleted();
    delete fqFileReader;
    std::cerr << "file " << file << " has " << n_chunks << " chunks" << std::endl;
    return 0;
}


__device__ inline randstrobe_hash_t gpu_get_hash(const RefRandstrobe *d_randstrobes, size_t d_randstrobes_size, my_bucket_index_t position) {
    if (position < d_randstrobes_size) {
        return d_randstrobes[position].hash;
    } else {
        return static_cast<randstrobe_hash_t>(-1);
    }
}

__device__ inline bool gpu_is_filtered(const RefRandstrobe *d_randstrobes, size_t d_randstrobes_size, my_bucket_index_t position,
                unsigned int filter_cutoff) {
    return gpu_get_hash(d_randstrobes, d_randstrobes_size, position) ==
           gpu_get_hash(d_randstrobes, d_randstrobes_size, position + filter_cutoff);
}

__device__ int gpu_get_count(
        const RefRandstrobe *d_randstrobes,
        const my_bucket_index_t *d_randstrobe_start_indices,
        my_bucket_index_t position,
        int bits
) {
    const auto key = d_randstrobes[position].hash;
    const unsigned int top_N = key >> (64 - bits);
    int64_t position_end = d_randstrobe_start_indices[top_N + 1];
    int64_t position_start = position;

    if(position_end == 0) return 0;
    int64_t low = position_start, high = position_end - 1, ans = 0;
    while (low <= high) {
        int64_t mid = (low + high) / 2;
        if (d_randstrobes[mid].hash == key) {
            low = mid + 1;
            ans = mid;
        } else {
            high = mid - 1;
        }
    }
    // int count = 0;
    // for (auto i = position_start; i < position_end; ++i) {
    //     if (d_randstrobes[i].hash == key){
    //         count += 1;
    //     } else break;
    // }
    // if(count != ans - position_start + 1) {
    //     // print d_randstrobes[position_start - position_end]
    //     // for(int i = position_start; i < position_end; i++) {
    //     //     printf("hash %llu ", d_randstrobes[i].hash);
    //     // }
    //     // printf("\n");
    //     printf("count %d %llu\n", count, ans - position_start + 1);
    // }

    return ans - position_start + 1;
}

// __device__ int gpu_get_count(
//         const RefRandstrobe *d_randstrobes,
//         const my_bucket_index_t *d_randstrobe_start_indices,
//         my_bucket_index_t position,
//         int bits

// )  {
//     const auto key = d_randstrobes[position].hash;
//     const unsigned int top_N = key >> (64 - bits);
//     my_bucket_index_t position_end = d_randstrobe_start_indices[top_N + 1];
//     // change the following for-loop to binary search

//     int count = 1;
//     for (my_bucket_index_t position_start = position + 1; position_start < position_end; ++position_start) {
//         if (d_randstrobes[position_start].hash == key){
//             count += 1;
//         } else break;
//     }
//     return count;
// }

__device__ inline size_t gpu_find(
        const RefRandstrobe *d_randstrobes,
        const my_bucket_index_t *d_randstrobe_start_indices,
        const randstrobe_hash_t key,
        int bits
) {
    const unsigned int top_N = key >> (64 - bits);
    my_bucket_index_t position_start = d_randstrobe_start_indices[top_N];
    my_bucket_index_t position_end = d_randstrobe_start_indices[top_N + 1];
    if(position_end - position_start < 64) {
        for (my_bucket_index_t i = position_start; i < position_end; ++i) {
            if (d_randstrobes[i].hash == key) {
                return i;
            }
        }
        return static_cast<size_t>(-1); // No match
    } else {
        my_bucket_index_t low = position_start, high = position_end;
        while (low < high) {
            my_bucket_index_t mid = low + (high - low) / 2;
            if (d_randstrobes[mid].hash < key) {
                low = mid + 1;
            } else {
                high = mid;
            }
        }
        if (d_randstrobes[low].hash == key) {
            return low;
        } else return static_cast<size_t>(-1); // No match
    }
}

template <typename T>
__device__ int partition(T* data, int low, int high) {
    T pivot = data[high];
    int i = low - 1;

    for (int j = low; j < high; ++j) {
        if (data[j] < pivot) {
            ++i;
            T temp = data[i];
            data[i] = data[j];
            data[j] = temp;
        }
    }

    T temp = data[i + 1];
    data[i + 1] = data[high];
    data[high] = temp;

    return i + 1;
}

template <typename T>
__device__ void dfs_quick_sort(T* data, int low, int high) {
    if (low < high) {
        int pivot_index = partition(data, low, high);

        // Recursively sort elements before and after the pivot
        dfs_quick_sort(data, low, pivot_index - 1);
        dfs_quick_sort(data, pivot_index + 1, high);
    }
}

template <typename T>
struct DefaultCompare {
    __device__ __forceinline__
    bool operator()(const T& a, const T& b) const {
        return a < b;
    }
};

//template <typename T, typename Compare = DefaultCompare<T>>
//__device__ void quick_sort_iterative(
//        T* data,
//        int low,
//        int high,
//        Compare comp = DefaultCompare<T>()
//) {
////    static_assert(
////            __is_invocable_bool<Compare, T, T>::value,
////            "Invalid comparison function signature"
////    );
//
//    int stack[64];
//    int top = -1;
//
//    stack[++top] = low;
//    stack[++top] = high;
//
//    while (top >= 0) {
//        high = stack[top--];
//        low = stack[top--];
//
//        if (high - low < 32) {
//            insertion_sort(data, low, high, comp);
//            continue;
//        }
//
//        int mid = low + (high - low)/2;
//        if (comp(data[high], data[mid])) my_swap(data[mid], data[high]);
//        if (comp(data[high], data[low])) my_swap(data[low], data[high]);
//        if (comp(data[mid], data[low])) my_swap(data[low], data[mid]);
//
//        T pivot = data[mid];
//
//        int i = low - 1;
//        int j = high + 1;
//        while (true) {
//            do { ++i; } while (comp(data[i], pivot));
//            do { --j; } while (comp(pivot, data[j]));
//            if (i >= j) break;
//            my_swap(data[i], data[j]);
//        }
//        int p = j;
//
//        if (p - low < high - p) {
//            if (low < p) {
//                stack[++top] = low;
//                stack[++top] = p;
//            }
//            if (p+1 < high) {
//                stack[++top] = p+1;
//                stack[++top] = high;
//            }
//        } else {
//            if (p+1 < high) {
//                stack[++top] = p+1;
//                stack[++top] = high;
//            }
//            if (low < p) {
//                stack[++top] = low;
//                stack[++top] = p;
//            }
//        }
//    }
//}
//
//
//template <typename T, typename Compare>
//__device__ void insertion_sort(T* data, int low, int high, Compare comp) {
//    for (int i = low + 1; i <= high; ++i) {
//        T key = data[i];
//        int j = i - 1;
//
//#pragma unroll
//        while (j >= low && comp(key, data[j])) {
//            data[j + 1] = data[j];
//            --j;
//        }
//        data[j + 1] = key;
//    }
//}

template <typename T, typename Compare = DefaultCompare<T>>
__device__ void quick_sort_iterative(
        T* data,
        int low,
        int high,
        Compare comp = DefaultCompare<T>()
) {
    if (low > high) return;
    int vec_size = high - low + 1;
    my_vector<int>stack_vec(vec_size * 2);
    int* stack = stack_vec.data;
    int top = -1;
    stack[++top] = low;
    stack[++top] = high;
    while (top >= 0) {
        high = stack[top--];
        low = stack[top--];
        // Partition
        T pivot = data[high];
        int i = low - 1;
        for (int j = low; j < high; ++j) {
            //if (data[j] < pivot) {
            if (comp(data[j], pivot)) {
                ++i;
                T temp = data[i];
                data[i] = data[j];
                data[j] = temp;
            }
        }
        T temp = data[i + 1];
        data[i + 1] = data[high];
        data[high] = temp;
        int pivot_index = i + 1;
        if (pivot_index - 1 > low) {
            stack[++top] = low;
            stack[++top] = pivot_index - 1;
        }
        if (pivot_index + 1 < high) {
            stack[++top] = pivot_index + 1;
            stack[++top] = high;
        }
    }
}


template <typename T>
__device__ void bubble_sort(T* data, int size) {
    for (int i = 0; i < size - 1; ++i) {
        for (int j = 0; j < size - i - 1; ++j) {
            if (data[j + 1] < data[j]) {
                T temp = data[j];
                data[j] = data[j + 1];
                data[j + 1] = temp;
            }
        }
    }
}

template <typename T>
__device__ void quick_sort(T* data, int size) {
    quick_sort_iterative(data, 0, size - 1);
//    bubble_sort(data, size);

}

struct Rescue_Seeds {
    int read_id;
    int read_fr;
    int seeds_num;
    QueryRandstrobe* seeds;
};

__device__ void print_nam(Nam nam) {
    printf("nam_id: %d, ref_id: %d, ref_start: %d, ref_end: %d, query_start: %d, query_end: %d, n_hits: %d, is_rc: %d\n",
           nam.nam_id, nam.ref_id, nam.ref_start, nam.ref_end, nam.query_start, nam.query_end, nam.n_hits, nam.is_rc);
}

__device__ void print_str(my_string str) {
    for(int i = 0; i < str.size(); i++) {
        printf("%c", str[i]);
    }
    printf("\n");
}

struct GPURead {
    char* seq;
    char* rc;
    int length;
    __device__ int size() const { return length; }
};

struct GPUAlignment {
    int ref_id;
    int ref_start;
    //my_vector<uint32_t> cigar;
    int edit_distance;
    int global_ed;
    int score;
    int length;
    bool is_rc;
    bool is_unaligned{false};
    // Whether a gapped alignment function was used to obtain this alignment
    // (even if true, the alignment can still be without gaps)
    bool gapped{false};
    //__device__ void move_to(GPUAlignment& dest) {
    //    dest.ref_id = ref_id;
    //    dest.ref_start = ref_start;
    //    //dest.cigar.move_from(cigar);
    //    dest.edit_distance = edit_distance;
    //    dest.global_ed = global_ed;
    //    dest.score = score;
    //    dest.length = length;
    //    dest.is_rc = is_rc;
    //    dest.is_unaligned = is_unaligned;
    //    dest.gapped = gapped;
    //}

};

struct GPUAlignTmpRes {
    int type;
    // type 0 : size1 == 0 size2 == 0, unmapped_pair
    // type 1 : size1 == 0, rescue read1
    // type 2 : size2 == 0, rescue read2
    // type 3 : good pair
    // type 4 : for loop
    int mapq1;
    int mapq2;
    int type4_loop_size;
    my_vector<bool> is_extend_seed;
    my_vector<bool> consistent_nam;
    my_vector<bool> is_read1;
    my_vector<Nam> type4_nams;
    my_vector<Nam> todo_nams;
    my_vector<bool> done_align;
    // if done_align, align_res is the alignment results
    my_vector<GPUAlignment> align_res;
    __device__ void init() {
        type = 0;
        mapq1 = 0;
        mapq2 = 0;
        type4_loop_size = 0;
    }
};

struct GPUReferences {
    my_vector<my_string> sequences;
    my_vector<size_t> lengths;
    int num_refs;
};

struct GPUAlignmentInfo {
    my_vector<uint32_t> cigar;
    unsigned int edit_distance{0};
    unsigned int ref_start{0};
    unsigned int ref_end{0};
    unsigned int query_start{0};
    unsigned int query_end{0};
    int sw_score{0};

    __device__ int ref_span() const { return ref_end - ref_start; }
};

__device__ void cigar_push(my_vector<uint32_t>& m_ops, uint8_t op, int len) {
    assert(op < 16);
    if (m_ops.empty() || (m_ops.back() & 0xf) != op) {
        m_ops.push_back(len << 4 | op);
    } else {
        m_ops.back() += len << 4;
    }
}

__device__ void my_hamming_align(const my_string &query, const my_string &ref, int match, int mismatch, int end_bonus, GPUAlignmentInfo& aln) {
    if (query.length() != ref.length()) {
        return;
    }
    size_t n = query.length();

    size_t start = 0; // start of the current segment
    int score = end_bonus; // accumulated score so far in the current segment

    size_t best_start = 0;
    size_t best_end = 0;
    int best_score = 0;
    for (size_t i = 0; i < n; ++i) {
        if (query[i] == ref[i]) {
            score += match;
        } else {
            score -= mismatch;
        }
        if (score < 0) {
            start = i + 1;
            score = 0;
        }
        if (score > best_score) {
            best_start = start;
            best_score = score;
            best_end = i + 1;
        }
    }
    if (score + end_bonus > best_score) {
        best_score = score + end_bonus;
        best_end = query.length();
        best_start = start;
    }

    size_t segment_start = best_start;
    size_t segment_end = best_end;
    score = best_score;

    if (segment_start > 0) {
        cigar_push(aln.cigar, CIGAR_SOFTCLIP, segment_start);
    }

    // Create CIGAR string and count mismatches
    int counter = 0;
    bool prev_is_match = false;
    int mismatches = 0;
    bool first = true;
    for (size_t i = segment_start; i < segment_end; i++) {
        bool is_match = query[i] == ref[i];
        mismatches += is_match ? 0 : 1;
        if (!first && is_match != prev_is_match) {
            cigar_push(aln.cigar, prev_is_match ? CIGAR_EQ : CIGAR_X, counter);
            counter = 0;
        }
        counter++;
        prev_is_match = is_match;
        first = false;
    }
    if (!first) {
        cigar_push(aln.cigar, prev_is_match ? CIGAR_EQ : CIGAR_X, counter);
    }

    int soft_right = query.length() - segment_end;
    if (soft_right > 0) {
        cigar_push(aln.cigar, CIGAR_SOFTCLIP, soft_right);
    }

    aln.sw_score = score;
    aln.edit_distance = mismatches;
    aln.ref_start = segment_start;
    aln.ref_end = segment_end;
    aln.query_start = segment_start;
    aln.query_end = segment_end;
    return;
}

__device__ bool extend_seed_part(
        GPUAlignTmpRes& align_tmp_res,
        const AlignmentParameters& aligner_parameters,
        const Nam& nam,
        const GPUReferences& references,
        const GPURead& read,
        bool consistent_nam
) {
    const my_string query(nam.is_rc ? read.rc : read.seq, read.length);
    const my_string ref = references.sequences[nam.ref_id];

    const auto projected_ref_start = my_max(0, nam.ref_start - nam.query_start);
    const auto projected_ref_end = my_min(nam.ref_end + query.size() - nam.query_end, ref.size());

    GPUAlignmentInfo info;
    int result_ref_start;
    bool gapped = true;
    if (projected_ref_end - projected_ref_start == query.size() && consistent_nam) {
        my_string ref_segm_ham = ref.substr(projected_ref_start, query.size());
        int hamming_dist = 0;
        for (size_t i = 0; i < query.size(); ++i) {
            if (query[i] != ref_segm_ham[i]) {
                ++hamming_dist;
            }
        }
        if (hamming_dist >= 0 && (((float) hamming_dist / query.size()) < 0.05)) {  //Hamming distance worked fine, no need to ksw align
            my_hamming_align(
                    query, ref_segm_ham, aligner_parameters.match, aligner_parameters.mismatch,
                    aligner_parameters.end_bonus, info
            );
            result_ref_start = projected_ref_start + info.ref_start;
            gapped = false;
        }
    }

    align_tmp_res.todo_nams.push_back(nam);
    align_tmp_res.is_extend_seed.push_back(true);
    if (gapped) {
        // not pass hamming, pending to do align on GPU, tag is false
        GPUAlignment alignment;
        align_tmp_res.done_align.push_back(false);
        align_tmp_res.align_res.push_back(alignment);
    } else {
        // pass hamming, store result, tag is true
        align_tmp_res.done_align.push_back(true);
        int softclipped = info.query_start + (query.size() - info.query_end);
        GPUAlignment alignment;
        //alignment.cigar.move_from(info.cigar);
        alignment.edit_distance = info.edit_distance;
        alignment.global_ed = info.edit_distance + softclipped;
        alignment.score = info.sw_score;
        alignment.ref_start = result_ref_start;
        alignment.length = info.ref_span();
        alignment.is_rc = nam.is_rc;
        alignment.is_unaligned = false;
        alignment.ref_id = nam.ref_id;
        alignment.gapped = gapped;
        align_tmp_res.align_res.push_back(alignment);
    }
    return gapped;
}

__device__ bool has_shared_substring(const my_string& read_seq, const my_string& ref_seq, int k) {
    int sub_size = 2 * k / 3;
    int step_size = k / 3;
    my_string submer;
    for (size_t i = 0; i + sub_size < read_seq.size(); i += step_size) {
        submer = read_seq.substr(i, sub_size);
        if (ref_seq.find(submer) != -1) {
            return true;
        }
    }
    return false;
}

__device__ bool rescue_mate_part(
        GPUAlignTmpRes& align_tmp_res,
        const AlignmentParameters& aligner_parameters,
        const Nam& nam,
        const GPUReferences& references,
        const GPURead& read,
        float mu,
        float sigma,
        int k
) {
    GPUAlignment alignment;
    int a, b;
    my_string r_tmp;
    auto read_len = read.size();

    if (nam.is_rc) {
        r_tmp = my_string(read.seq, read_len);
        a = nam.ref_start - nam.query_start - (mu + 5 * sigma);
        b = nam.ref_start - nam.query_start + read_len / 2;  // at most half read overlap
    } else {
        r_tmp = my_string(read.rc, read_len);                                             // mate is rc since fr orientation
        a = nam.ref_end + (read_len - nam.query_end) - read_len / 2;  // at most half read overlap
        b = nam.ref_end + (read_len - nam.query_end) + (mu + 5 * sigma);
    }

    auto ref_len = static_cast<int>(references.lengths[nam.ref_id]);
    auto ref_start = my_max(0, my_min(a, ref_len));
    auto ref_end = my_min(ref_len, my_max(0, b));

    align_tmp_res.todo_nams.push_back(nam);
    align_tmp_res.is_extend_seed.push_back(false);
    if (ref_end < ref_start + k) {
//        alignment.cigar = Cigar();
        alignment.edit_distance = read_len;
        alignment.score = 0;
        alignment.ref_start = 0;
        alignment.is_rc = nam.is_rc;
        alignment.ref_id = nam.ref_id;
        alignment.is_unaligned = true;
        align_tmp_res.done_align.push_back(true);
        align_tmp_res.align_res.push_back(alignment);
        return true;
    }
    my_string ref_segm = references.sequences[nam.ref_id].substr(ref_start, ref_end - ref_start);

    if (!has_shared_substring(r_tmp, ref_segm, k)) {
//        alignment.cigar = Cigar();
        alignment.edit_distance = read_len;
        alignment.score = 0;
        alignment.ref_start = 0;
        alignment.is_rc = nam.is_rc;
        alignment.ref_id = nam.ref_id;
        alignment.is_unaligned = true;
        align_tmp_res.done_align.push_back(true);
        align_tmp_res.align_res.push_back(alignment);
        return true;
    }

    align_tmp_res.done_align.push_back(false);
    align_tmp_res.align_res.push_back(alignment);
    return false;
}


__device__ bool reverse_nam_if_needed(Nam& nam, const GPURead& read, const GPUReferences& references, int k) {
    auto read_len = read.size();
    my_string ref_start_kmer = references.sequences[nam.ref_id].substr(nam.ref_start, k);
    my_string ref_end_kmer = references.sequences[nam.ref_id].substr(nam.ref_end - k, k);


    my_string seq, seq_rc;
    if (nam.is_rc) {
        seq = my_string(read.rc, read_len);
        seq_rc = my_string(read.seq, read_len);
    } else {
        seq = my_string(read.seq, read_len);
        seq_rc = my_string(read.rc, read_len);
    }
    my_string read_start_kmer = seq.substr(nam.query_start, k);
    my_string read_end_kmer = seq.substr(nam.query_end - k, k);
    if (ref_start_kmer == read_start_kmer && ref_end_kmer == read_end_kmer) {
        return true;
    }

    // False forward or false reverse (possible due to symmetrical hash values)
    //    we need two extra checks for this - hopefully this will remove all the false hits we see (true hash collisions should be very few)
    int q_start_tmp = read_len - nam.query_end;
    int q_end_tmp = read_len - nam.query_start;
    // false reverse hit, change coordinates in nam to forward
    read_start_kmer = seq_rc.substr(q_start_tmp, k);
    read_end_kmer = seq_rc.substr(q_end_tmp - k, k);
    if (ref_start_kmer == read_start_kmer && ref_end_kmer == read_end_kmer) {
        nam.is_rc = !nam.is_rc;
        nam.query_start = q_start_tmp;
        nam.query_end = q_end_tmp;
        return true;
    }
    return false;
}

__device__ void rescue_read_part(
        int flag,
        GPUAlignTmpRes& align_tmp_res,
        const GPURead& read2,  // read to be rescued
        const GPURead& read1,  // read that has NAMs
        const AlignmentParameters& aligner_parameters,
        const GPUReferences& references,
        my_vector<Nam>& nams1,
        int max_tries,
        float dropoff,
        int k,
        float mu,
        float sigma,
        size_t max_secondary,
        double secondary_dropoff,
        bool swap_r1r2
) {
    //align_tmp_res.type = flag;
    Nam n_max1 = nams1[0];
    int tries = 0;
    // this loop is safe, loop size is stable
    for (int i = 0; i < nams1.size(); i++) {
        Nam &nam = nams1[i];
        float score_dropoff1 = (float) nam.n_hits / n_max1.n_hits;
        // only consider top hits (as minimap2 does) and break if below dropoff cutoff.
        if (tries >= max_tries || score_dropoff1 < dropoff) {
            break;
        }

        const bool consistent_nam = reverse_nam_if_needed(nam, read1, references, k);
        // reserve extend and store info
        if(flag == 1) align_tmp_res.is_read1.push_back(true);
        else align_tmp_res.is_read1.push_back(false);
        bool gapped = extend_seed_part(align_tmp_res, aligner_parameters, nam, references, read1, consistent_nam);

        // Force SW alignment to rescue mate
        if(flag == 1) align_tmp_res.is_read1.push_back(false);
        else align_tmp_res.is_read1.push_back(true);
        bool is_unaligned = rescue_mate_part(align_tmp_res, aligner_parameters, nam, references, read2, mu, sigma, k);
        tries++;
    }
}

__device__ bool is_proper_nam_pair(const Nam nam1, const Nam nam2, float mu, float sigma) {
    if (nam1.ref_id != nam2.ref_id || nam1.is_rc == nam2.is_rc) {
        return false;
    }
    int a = my_max(0, nam1.ref_start - nam1.query_start);
    int b = my_max(0, nam2.ref_start - nam2.query_start);

    // r1 ---> <---- r2
    bool r1_r2 = nam2.is_rc && (a <= b) && (b - a < mu + 10 * sigma);
    if(r1_r2) return 1;

    // r2 ---> <---- r1
    bool r2_r1 = nam1.is_rc && (b <= a) && (a - b < mu + 10 * sigma);
    if(r2_r1) return 1;
    return 0;

//    return r1_r2 || r2_r1;
}

__device__ float top_dropoff(my_vector<Nam>& nams) {
    auto& n_max = nams[0];
    if (n_max.n_hits <= 2) {
        return 1.0;
    }
    if (nams.size() > 1) {
        return (float) nams[1].n_hits / n_max.n_hits;
    }
    return 0.0;
}

__device__ uint8_t get_mapq(const my_vector<Nam>& nams, const Nam& n_max) {
    if (nams.size() <= 1) {
        return 60;
    }
    const float s1 = n_max.score;
    const float s2 = nams[1].score;
    // from minimap2: MAPQ = 40(1−s2/s1) ·min{1,|M|/10} · log s1
    const float min_matches = my_min(n_max.n_hits / 10.0, 1.0);
    const int uncapped_mapq = 40 * (1 - s2 / s1) * min_matches * log(s1);
    return my_min(uncapped_mapq, 60);
}

__device__ bool is_proper_pair(const GPUAlignment& alignment1, const GPUAlignment& alignment2, float mu, float sigma) {
    const int dist = alignment2.ref_start - alignment1.ref_start;
    const bool same_reference = alignment1.ref_id == alignment2.ref_id;
    const bool both_aligned = same_reference && !alignment1.is_unaligned && !alignment2.is_unaligned;
    const bool r1_r2 = !alignment1.is_rc && alignment2.is_rc && dist >= 0; // r1 ---> <---- r2
    const bool r2_r1 = !alignment2.is_rc && alignment1.is_rc && dist <= 0; // r2 ---> <---- r1
    const bool rel_orientation_good = r1_r2 || r2_r1;
    const bool insert_good = std::abs(dist) <= mu + 6 * sigma;

    return both_aligned && insert_good && rel_orientation_good;
}

struct GPUInsertSizeDistribution {
    float sample_size = 1;
    float mu = 300;
    float sigma = 100;
    float V = 10000;
    float SSE = 10000;

    // Add a new observation
    __device__ void update(int dist) {
        if (dist >= 2000) {
            return;
        }
        const float e = dist - mu;
        mu += e / sample_size;  // (1.0/(sample_size +1.0)) * (sample_size*mu + d);
        SSE += e * (dist - mu);
        if (sample_size > 1) {
            //d < 1000 ? ((sample_size +1.0)/sample_size) * ( (V*sample_size/(sample_size +1)) + ((mu-d)*(mu-d))/sample_size ) : V;
            V = SSE / (sample_size - 1.0);
        } else {
            V = SSE;
        }
        sigma = sqrtf(V);
        sample_size = sample_size + 1.0;
        if (mu < 0) {
            printf("mu negative, mu: %f sigma: %f SSE: %f sample size: %f\n", mu, sigma, SSE, sample_size);
            assert(false);
        }
        if (SSE < 0) {
            printf("SSE negative, mu: %f sigma: %f SSE: %f sample size: %f\n", mu, sigma, SSE, sample_size);
            assert(false);
        }
    }
};


struct NamPair {
    int score;
    Nam nam1;
    Nam nam2;
};

__device__ my_vector<NamPair> get_best_scoring_nam_pairs(
        const my_vector<Nam>& nams1,
        const my_vector<Nam>& nams2,
        float mu,
        float sigma
) {
    my_vector<NamPair> joint_nam_scores;
    if (nams1.empty() && nams2.empty()) {
        return joint_nam_scores;
    }

    my_vector<bool> added_n1(nams1.size());
    my_vector<bool> added_n2(nams2.size());
    for(int i = 0; i < nams1.size(); i++) added_n1[i] = false;
    for(int i = 0; i < nams2.size(); i++) added_n2[i] = false;

    int best_joint_hits = 0;

    my_vector<Nam> nams2_sorted[2];
    for(int i = 0; i < nams2.size(); i++) {
        Nam nam2 = nams2[i];
        nams2_sorted[nam2.is_rc].push_back(nam2);
    }

    for(int id = 0; id < 2; id++) {
        quick_sort_iterative(&(nams2_sorted[id][0]), 0, nams2_sorted[id].size() - 1, [](const Nam &n1, const Nam &n2) {
            int val1 = my_max(0, n1.ref_start - n1.query_start);
            int val2 = my_max(0, n2.ref_start - n2.query_start);
//            if (val1 == val2) {
//                return n1.nam_id < n2.nam_id;
//            } else {
                return val1 < val2;
//            }
        });

//        Nam* data = &(nams2_sorted[id][0]);
//        int size = nams2_sorted[id].size();
//        for (int i = 0; i < size - 1; ++i) {
//            for (int j = 0; j < size - i - 1; ++j) {
//                Nam n1 = data[j];
//                Nam n2 = data[j + 1];
//                int val1 = my_max(0, n1.ref_start - n1.query_start);
//                int val2 = my_max(0, n2.ref_start - n2.query_start);
//                if (val1 > val2) {
//                    Nam temp = data[j];
//                    data[j] = data[j + 1];
//                    data[j + 1] = temp;
//                }
//            }
//        }

        // check if sort res is ok
        for (int i = 0; i < nams2_sorted[id].size(); i++) {
            Nam nam2 = nams2_sorted[id][i];
            int val1 = my_max(0, nam2.ref_start - nam2.query_start);
            if (i > 0) {
                Nam nam2_prev = nams2_sorted[id][i - 1];
                int val2 = my_max(0, nam2_prev.ref_start - nam2_prev.query_start);
                if (val1 < val2) {
                    printf("sort error: %d %d %d %d\n", i, val1, val2, id);
                    assert(false);
                }
            }
        }

    }

    for(int i = 0; i < nams1.size(); i++) {
        Nam nam1 = nams1[i];
        int nam1_val = my_max(0, nam1.ref_start - nam1.query_start);
        if(nam1.is_rc == 1) {
            float L_val = nam1_val - (mu + 10 * sigma);
            float R_val = nam1_val;
            int ll = 0, rr = nams2_sorted[0].size() - 1, ans_pos = nams2_sorted[0].size();
            while(ll <= rr) {
                int mid  = (ll + rr) / 2;
                int now_val = my_max(0, nams2_sorted[0][mid].ref_start - nams2_sorted[0][mid].query_start);
                if(now_val > L_val) {
                    rr = mid - 1;
                    ans_pos = mid;
                } else {
                    ll = mid + 1;
                }
            }

            for (int id = ans_pos; id < nams2_sorted[0].size(); id++) {
                Nam nam2 = nams2_sorted[0][id];
                int joint_hits = nam1.n_hits + nam2.n_hits;
                if(nam1.ref_id != nam2.ref_id) continue;
                int a = my_max(0, nam1.ref_start - nam1.query_start);
                int b = my_max(0, nam2.ref_start - nam2.query_start);
                if(b > R_val - 1e-6) break;

                // nam1 is rec, nam2 is fwd

                //// r1 ---> <---- r2
                //bool r1_r2 = nam2.is_rc && (a <= b) && (b - a < mu + 10 * sigma);

                // r2 ---> <---- r1
                bool r2_r1 = (a - b >= 0) && (a - b < mu + 10 * sigma);

                if (r2_r1) {
                    joint_nam_scores.push_back(NamPair{joint_hits, nam1, nam2});
                    added_n1[nam1.nam_id] = 1;
                    added_n2[nam2.nam_id] = 1;
                    //best_joint_hits = my_max(joint_hits, best_joint_hits);
                }
            }
        } else{
            float L_val = nam1_val;
            float R_val = nam1_val + mu + 10 * sigma;
            int ll = 0, rr = nams2_sorted[1].size() - 1, ans_pos = nams2_sorted[1].size();
            while(ll <= rr) {
                int mid  = (ll + rr) / 2;
                int now_val = my_max(0, nams2_sorted[1][mid].ref_start - nams2_sorted[1][mid].query_start);
                if(now_val >= L_val) {
                    rr = mid - 1;
                    ans_pos = mid;
                } else {
                    ll = mid + 1;
                }
            }

            for (int id = ans_pos; id < nams2_sorted[1].size(); id++) {
                Nam nam2 = nams2_sorted[1][id];
                int joint_hits = nam1.n_hits + nam2.n_hits;
                if(nam1.ref_id != nam2.ref_id) continue;

                int a = my_max(0, nam1.ref_start - nam1.query_start);
                int b = my_max(0, nam2.ref_start - nam2.query_start);
                if(b >= R_val - 1e-6) break;

                // nam1 is fwd, nam2 is rec

                // r1 ---> <---- r2
                bool r1_r2 = (b - a >= 0) && (b - a < mu + 10 * sigma);

                if (r1_r2) {
                    joint_nam_scores.push_back(NamPair{joint_hits, nam1, nam2});
                    added_n1[nam1.nam_id] = 1;
                    added_n2[nam2.nam_id] = 1;
                    //best_joint_hits = my_max(joint_hits, best_joint_hits);
                }
            }
        }

    }

//    printf("joint_nam_scores : %d\n", joint_nam_scores.size());



    // Find high-scoring R1 NAMs that are not part of a proper pair
    Nam dummy_nam;
    dummy_nam.ref_start = -1;
    if (!nams1.empty()) {
        int best_joint_hits1 = best_joint_hits > 0 ? best_joint_hits : nams1[0].n_hits;
        for(int i = 0; i < nams1.size(); i++) {
            Nam nam1 = nams1[i];
            if (nam1.n_hits < best_joint_hits1 / 2) {
                break;
            }
            if (added_n1[nam1.nam_id]) {
                continue;
            }
            joint_nam_scores.push_back(NamPair{nam1.n_hits, nam1, dummy_nam});
        }
    }


    // Find high-scoring R2 NAMs that are not part of a proper pair
    if (!nams2.empty()) {
        int best_joint_hits2 = best_joint_hits > 0 ? best_joint_hits : nams2[0].n_hits;
        for(int i = 0; i < nams2.size(); i++) {
            Nam nam2 = nams2[i];
            if (nam2.n_hits < best_joint_hits2 / 2) {
                break;
            }
            if (added_n2[nam2.nam_id]) {
                continue;
            }
            joint_nam_scores.push_back(NamPair{nam2.n_hits, dummy_nam, nam2});
        }
    }



    quick_sort_iterative(&(joint_nam_scores[0]), 0, joint_nam_scores.size() - 1, [](const NamPair &n1, const NamPair &n2) {
//        if (n1.score == n2.score) {
//            if (n1.nam1.nam_id == n2.nam1.nam_id) {
//                return n1.nam2.nam_id < n2.nam2.nam_id;
//            } else {
//                return n1.nam1.nam_id < n2.nam1.nam_id;
//            }
//        } else {
            return n1.score > n2.score;
//        }
    });

//    NamPair* data = &(joint_nam_scores[0]);
//    int size = joint_nam_scores.size();
//    for (int i = 0; i < size - 1; ++i) {
//        for (int j = 0; j < size - i - 1; ++j) {
//            NamPair n1 = data[j];
//            NamPair n2 = data[j + 1];
//            if (n1.score < n2.score) {
//                NamPair temp = data[j];
//                data[j] = data[j + 1];
//                data[j + 1] = temp;
//            }
//        }
//    }
    // check if sort res is ok
    for (int i = 0; i < joint_nam_scores.size(); i++) {
        NamPair nam2 = joint_nam_scores[i];
        if (i > 0) {
            NamPair nam2_prev = joint_nam_scores[i - 1];
            if (nam2.score > nam2_prev.score) {
                printf("sort error: %d %d %d\n", i, nam2.score, nam2_prev.score);
                assert(false);
            }
        }
    }

    return joint_nam_scores;
}

__device__ static unsigned char revcomp_table[256] = {
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'T', 'N', 'G',  'N', 'N', 'N', 'C',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'A', 'A', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'T', 'N', 'G',  'N', 'N', 'N', 'C',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'A', 'A', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
        'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N'
};

struct GPUScoredAlignmentPair {
    double score;
    GPUAlignment alignment1;
    GPUAlignment alignment2;
};



__device__ void align_PE_part(
        GPUAlignTmpRes& align_tmp_res,
        const AlignmentParameters& aligner_parameters,
        my_vector<Nam>& nams1,
        my_vector<Nam>& nams2,
        char* seq1, int seq_len1,
        char* seq2, int seq_len2,
        int k,
        const GPUReferences& references,
        float dropoff,
        GPUInsertSizeDistribution& isize_est,
        unsigned max_tries,
        size_t max_secondary
) {
    const auto mu = isize_est.mu;
    const auto sigma = isize_est.sigma;
    char* rc1 = (char*)my_malloc(seq_len1 + 1);
    char* rc2 = (char*)my_malloc(seq_len2 + 1);
    for(int i = 0; i < seq_len1; i++) {
        rc1[i] = revcomp_table[static_cast<int>(seq1[seq_len1 - i - 1])];
    }
    for(int i = 0; i < seq_len2; i++) {
        rc2[i] = revcomp_table[static_cast<int>(seq2[seq_len2 - i - 1])];
    }
    GPURead read1{seq1, rc1, seq_len1};
    GPURead read2{seq2, rc2, seq_len2};
    double secondary_dropoff = 2 * aligner_parameters.mismatch + aligner_parameters.gap_open;


    if (nams1.empty() && nams2.empty()) {
        // None of the reads have any NAMs
        align_tmp_res.type = 0;
        my_free(rc1);
        my_free(rc2);
        return;
    } 

    if (!nams1.empty() && nams2.empty()) {
        // Only read 1 has NAMS: attempt to rescue read 2
        align_tmp_res.type = 1;
        rescue_read_part(
                1, align_tmp_res, read2, read1, aligner_parameters, references, nams1, max_tries, dropoff, k, mu,
                sigma, max_secondary, secondary_dropoff, false
        );
        my_free(rc1);
        my_free(rc2);
        return;
    }

    if (nams1.empty() && !nams2.empty()) {
        // Only read 2 has NAMS: attempt to rescue read 1
        align_tmp_res.type = 2;
        rescue_read_part(
                2, align_tmp_res, read1, read2, aligner_parameters, references, nams2, max_tries, dropoff, k, mu,
                sigma, max_secondary, secondary_dropoff, true
        );
        my_free(rc1);
        my_free(rc2);
        return;
    }

    //    printf("type3\n");

    // If we get here, both reads have NAMs
    assert(!nams1.empty() && !nams2.empty());

    // Deal with the typical case that both reads map uniquely and form a proper pair
    if (top_dropoff(nams1) < dropoff && top_dropoff(nams2) < dropoff &&
        is_proper_nam_pair(nams1[0], nams2[0], mu, sigma)) {
        align_tmp_res.type = 3;
        Nam n_max1 = nams1[0];
        Nam n_max2 = nams2[0];

        bool consistent_nam1 = reverse_nam_if_needed(n_max1, read1, references, k);
        bool consistent_nam2 = reverse_nam_if_needed(n_max2, read2, references, k);

        align_tmp_res.is_read1.push_back(true);
        bool gapped1 = extend_seed_part(align_tmp_res, aligner_parameters, n_max1, references, read1, consistent_nam1);


        align_tmp_res.is_read1.push_back(false);
        bool gapped2 = extend_seed_part(align_tmp_res, aligner_parameters, n_max2, references, read2, consistent_nam2);

        int mapq1 = get_mapq(nams1, n_max1);
        int mapq2 = get_mapq(nams2, n_max2);
        align_tmp_res.mapq1 = mapq1;
        align_tmp_res.mapq2 = mapq2;


        if(!gapped1 && !gapped2) {
            int res_size = align_tmp_res.align_res.size();
            if(res_size < 2) {
                printf("align_tmp_res.align_res.size error %d\n", res_size);
                assert(false);
            }
            GPUAlignment alignment1 = align_tmp_res.align_res[res_size - 2];
            GPUAlignment alignment2 = align_tmp_res.align_res[res_size - 1];
            if(alignment1.gapped || alignment2.gapped) {
                printf("alignment gapped error\n");
                assert(false);
            }
            bool is_proper = is_proper_pair(alignment1, alignment2, mu, sigma);
            if ((isize_est.sample_size < 400) && (alignment1.edit_distance + alignment2.edit_distance < 3) && is_proper) {
                isize_est.update(my_abs(alignment1.ref_start - alignment2.ref_start));
            }
        }
        my_free(rc1);
        my_free(rc2);
        return;
    }

//    printf("type4\n");

    my_vector<NamPair> joint_nam_scores = get_best_scoring_nam_pairs(nams1, nams2, mu, sigma);
    align_tmp_res.type = 4 + joint_nam_scores.size();
    my_free(rc1);
    my_free(rc2);
    return;

    //if (threadIdx.x == 0 && blockIdx.x % 1000 == 0) printf("joint_nam_scores size %d\n", joint_nam_scores.size());
    my_vector<bool> is_aligned1(nams1.size());
    my_vector<bool> is_aligned2(nams2.size());
    for (int i = 0; i < nams1.size(); i++) is_aligned1[i] = 0;
    for (int i = 0; i < nams2.size(); i++) is_aligned2[i] = 0;

    {
        Nam n1_max = nams1[0];
        bool consistent_nam1 = reverse_nam_if_needed(n1_max, read1, references, k);
        align_tmp_res.is_read1.push_back(true);
        bool gapped1 = extend_seed_part(align_tmp_res, aligner_parameters, n1_max, references, read1, consistent_nam1);
        is_aligned1[n1_max.nam_id] = 1;

        Nam n2_max = nams2[0];
        bool consistent_nam2 = reverse_nam_if_needed(n2_max, read2, references, k);
        align_tmp_res.is_read1.push_back(false);
        bool gapped2 = extend_seed_part(align_tmp_res, aligner_parameters, n2_max, references, read2, consistent_nam2);
        is_aligned2[n2_max.nam_id] = 1;
    }

    // Turn pairs of high-scoring NAMs into pairs of alignments
    my_vector<GPUScoredAlignmentPair> high_scores;
    double max_score = joint_nam_scores[0].score;
    align_tmp_res.type4_loop_size = 0;
    for(int i = 0; i < joint_nam_scores.size(); i++) {
        double score_ = joint_nam_scores[i].score;
        Nam n1 = joint_nam_scores[i].nam1;
        Nam n2 = joint_nam_scores[i].nam2;
        float score_dropoff = (float) score_ / max_score;
        if (high_scores.size() >= max_tries || score_dropoff < dropoff) {
            break;
        }

        align_tmp_res.type4_nams.push_back(n1);
        align_tmp_res.type4_nams.push_back(n2);
        align_tmp_res.type4_loop_size++;

        if (n1.ref_start >= 0) {
            if (is_aligned1[n1.nam_id] == 1) {

            } else {
                bool consistent_nam = reverse_nam_if_needed(n1, read1, references, k);
                align_tmp_res.is_read1.push_back(true);
                bool gapped = extend_seed_part(align_tmp_res, aligner_parameters, n1, references, read1, consistent_nam);
                is_aligned1[n1.nam_id] = 1;
            }
        } else {
            reverse_nam_if_needed(n2, read2, references, k);
            align_tmp_res.is_read1.push_back(true);
            bool is_unaligned = rescue_mate_part(align_tmp_res, aligner_parameters, n2, references, read1, mu, sigma, k);

        }

        // ref_start == -1 is a marker for a dummy NAM
        if (n2.ref_start >= 0) {
            if (is_aligned2[n2.nam_id] == 1) {

            } else {
                bool consistent_nam = reverse_nam_if_needed(n2, read2, references, k);
                align_tmp_res.is_read1.push_back(false);
                bool gapped = extend_seed_part(align_tmp_res, aligner_parameters, n2, references, read2, consistent_nam);
                is_aligned2[n2.nam_id] = 1;
            }
        } else {
            reverse_nam_if_needed(n1, read1, references, k);
            align_tmp_res.is_read1.push_back(false);
            bool is_unaligned = rescue_mate_part(align_tmp_res, aligner_parameters, n1, references, read2, mu, sigma, k);
        }
        GPUScoredAlignmentPair aln_pair;
        high_scores.push_back(aln_pair);
    }

    my_free(rc1);
    my_free(rc2);
    return;
}

#define BLOCK_SIZE 32


__device__ void sort_hits_single(
        my_vector<my_pair<int, Hit>>& hits_per_ref
) {
    //bubble_sort(&(hits_per_ref[0]), hits_per_ref.size());
    quick_sort(&(hits_per_ref[0]), hits_per_ref.size());
}

struct ref_ids_edge {
    int pre;
    int ref_id;
};

#define key_mod_val 29

__device__ int find_ref_ids(int ref_id, int* head, ref_ids_edge* edges) {
    int key = ref_id % key_mod_val;
    for (int i = head[key]; i != -1; i = edges[i].pre) {
        if (edges[i].ref_id == ref_id) return i;
    }
    return -1;
}

__device__ void sort_hits_by_refid_get_ref_size(
        my_vector<my_pair<int, Hit>>& hits_per_ref,
        my_vector<my_pair<int, int>>& ref_size
) {
    my_vector<my_pair<int, my_vector<Hit>*>> all_hits;
    int *head = (int*)my_malloc(key_mod_val * sizeof(int));
    my_vector<ref_ids_edge> edges;
    for(int i = 0; i < key_mod_val; i++) head[i] = -1;
    int ref_ids_num = 0;
    for(int i = 0; i < hits_per_ref.size(); i++) {
        int ref_id = hits_per_ref[i].first;
        int find_ref_id_rank = find_ref_ids(ref_id, head, edges.data);
        if (find_ref_id_rank == -1) {
            find_ref_id_rank = ref_ids_num;
            int key = ref_id % key_mod_val;
            edges.push_back({head[key], ref_id});
            head[key] = ref_ids_num++;
            my_vector<Hit>* hits = (my_vector<Hit>*)my_malloc(sizeof(my_vector<Hit>));
            hits->init();
            all_hits.push_back({ref_id, hits});
        }
        all_hits[find_ref_id_rank].second->push_back(hits_per_ref[i].second);
    }
    hits_per_ref.clear();
    my_vector<my_pair<int, int>>* ref_sizes_ptr = (my_vector<my_pair<int, int>>*)my_malloc(sizeof(my_vector<my_pair<int, int>>));
    ref_sizes_ptr->init();
    for(int i = 0; i < all_hits.size(); i++) {
        ref_sizes_ptr->push_back({hits_per_ref.size(), all_hits[i].second->size()});
        for(int j = 0; j < all_hits[i].second->size(); j++) {
            hits_per_ref.push_back({all_hits[i].first, (*all_hits[i].second)[j]});
        }
        all_hits[i].second->release();
        my_free(all_hits[i].second);
    }
    ref_size = *ref_sizes_ptr;
    my_free(ref_sizes_ptr);
    my_free(head);
}

__device__ void sort_hits_by_refid(
        my_vector<my_pair<int, Hit>>& hits_per_ref
) {
    my_vector<my_pair<int, my_vector<Hit>*>> all_hits(32);
    int *head = (int*)my_malloc(key_mod_val * sizeof(int));
    my_vector<ref_ids_edge> edges(32);
    for(int i = 0; i < key_mod_val; i++) head[i] = -1;
    int ref_ids_num = 0;
    for(int i = 0; i < hits_per_ref.size(); i++) {
        int ref_id = hits_per_ref[i].first;
        int find_ref_id_rank = find_ref_ids(ref_id, head, edges.data);
        if (find_ref_id_rank == -1) {
            find_ref_id_rank = ref_ids_num;
            int key = ref_id % key_mod_val;
            edges.push_back({head[key], ref_id});
            head[key] = ref_ids_num++;
            my_vector<Hit>* hits = (my_vector<Hit>*)my_malloc(sizeof(my_vector<Hit>));
            hits->init(32);
            all_hits.push_back({ref_id, hits});
        }
        all_hits[find_ref_id_rank].second->push_back(hits_per_ref[i].second);
    }
    hits_per_ref.clear();
    for(int i = 0; i < all_hits.size(); i++) {
        for(int j = 0; j < all_hits[i].second->size(); j++) {
            hits_per_ref.push_back({all_hits[i].first, (*all_hits[i].second)[j]});
        }
        all_hits[i].second->release();
        my_free(all_hits[i].second);
    }
    my_free(head);
}

__device__ void sort_hits_parallel(
        my_vector<my_pair<int, Hit>>& hits_per_ref,
        int k,
        bool is_revcomp,
        int tid
) {
    if(hits_per_ref.size() == 0) return;
    int num_hits = hits_per_ref.size();

    const int items_per_thread = 160;
    int real_num_hits = items_per_thread * BLOCK_SIZE;
    if(real_num_hits < num_hits) {
        printf("real_num_hits %d num_hits %d\n", real_num_hits, num_hits);
    }
    assert(real_num_hits >= num_hits);

    typedef hipcub::BlockRadixSort<unsigned long long, BLOCK_SIZE, items_per_thread, int> BlockRadixSort;
    __shared__ typename BlockRadixSort::TempStorage temp_storage;

    unsigned long long thread_keys[items_per_thread];
    int thread_indices[items_per_thread];

    __shared__ int* old_ref_end;
    __shared__ int* old_query_end;
    if(tid == 0) {
        old_ref_end = (int*)my_malloc(real_num_hits * sizeof(int));
        old_query_end = (int*)my_malloc(real_num_hits * sizeof(int));
    }
    __syncthreads();

    for (int i = 0; i < items_per_thread; ++i) {
        int idx = tid * items_per_thread + i;
        if (idx < num_hits) {
            thread_keys[i] = (static_cast<unsigned long long>(hits_per_ref[idx].first) << 48) |
                             (static_cast<unsigned long long>(hits_per_ref[idx].second.query_start & 0xFFFF) << 32) |
                             (static_cast<unsigned long long>(hits_per_ref[idx].second.ref_start) & 0xFFFFFFFF);
            thread_indices[i] = idx;
            old_ref_end[idx] = hits_per_ref[idx].second.ref_end;
            old_query_end[idx] = hits_per_ref[idx].second.query_end;
        } else {
            thread_keys[i] = ULLONG_MAX;
            thread_indices[i] = -1;
            old_ref_end[idx] = 0;
            old_query_end[idx] = 0;
        }
    }
    __syncthreads();

    BlockRadixSort(temp_storage).Sort(thread_keys, thread_indices);
    __syncthreads();

    for (int i = 0; i < items_per_thread; ++i) {
        int idx = tid * items_per_thread + i;
        if (idx < num_hits) {
            hits_per_ref[idx].first = thread_keys[i] >> 48;
            hits_per_ref[idx].second.query_start = (thread_keys[i] >> 32) & 0xFFFF;
            hits_per_ref[idx].second.ref_start = thread_keys[i] & 0xFFFFFFFF;
            hits_per_ref[idx].second.ref_end = old_ref_end[thread_indices[i]];
            hits_per_ref[idx].second.query_end = old_query_end[thread_indices[i]];
        }
    }
    __syncthreads();
    if(tid == 0) {
        my_free(old_ref_end);
        my_free(old_query_end);
    }


}

__device__ size_t my_lower_bound(my_pair<int, Hit>* hits, size_t i_start, size_t i_end, int target) {
    size_t left = i_start, right = i_end;
    while (left < right) {
        size_t mid = left + (right - left) / 2;
        if (hits[mid].second.ref_start < target) {
            left = mid + 1;
        } else {
            right = mid;
        }
    }
    return left;
}

__device__ void salign_merge_hits(
        my_vector<my_pair<int, Hit>>& hits_per_ref,
        int k,
        bool is_revcomp,
        my_vector<Nam>& nams
) {
    if(hits_per_ref.size() == 0) return;
    int ref_num = 0;
    my_vector<int> each_ref_size;
    int pre_ref_id = hits_per_ref[0].first;
    int now_ref_num = 1;
    for(int i = 1; i < hits_per_ref.size(); i++) {
        int ref_id = hits_per_ref[i].first;
        Hit hit = hits_per_ref[i].second;
        if(ref_id != pre_ref_id) {
            ref_num++;
            pre_ref_id = ref_id;
            each_ref_size.push_back(now_ref_num);
            now_ref_num = 1;
        } else {
            now_ref_num++;
        }
    }
    ref_num++;
    each_ref_size.push_back(now_ref_num);
    //int mx_hits_per_ref = 0;
    //for (int i = 0; i < each_ref_size.size(); i++) {
    //    mx_hits_per_ref = my_max(mx_hits_per_ref, each_ref_size[i]);
    //}

    my_vector<Nam> open_nams;
    //(mx_hits_per_ref);

    int now_vec_pos = 0;
    for (int rid = 0; rid < ref_num; rid++) {
        if(rid != 0) now_vec_pos += each_ref_size[rid - 1];
        int ref_id = hits_per_ref[now_vec_pos].first;
        open_nams.clear();
        unsigned int prev_q_start = 0;
        size_t hits_size = each_ref_size[rid];
        my_pair<int, Hit>* hits = &(hits_per_ref[now_vec_pos]);
        for (size_t i = 0; i < hits_size; ) {
            size_t i_start = i;
            size_t i_end = i + 1;
            size_t i_size;
            while(i_end < hits_size && hits[i_end].second.query_start == hits[i].second.query_start) i_end++;
            i = i_end;
            i_size = i_end - i_start;
            //for(int j = 0; j < i_size - 1; j++) {
            //    assert(hits[i_start + j].second.ref_start <= hits[i_start + j + 1].second.ref_start);
            //}
            //quick_sort(&(hits[i_start]), i_size);
            my_vector<bool> is_added(i_size);
            for(size_t j = 0; j < i_size; j++) is_added.push_back(false);
            int query_start = hits[i_start].second.query_start;
            int cnt_done = 0;
            for (int k = 0; k < open_nams.size(); k++) {
                Nam& o = open_nams[k];
                if ( query_start > o.query_end ) continue;
                size_t lower = my_lower_bound(hits, i_start, i_end, o.ref_prev_hit_startpos + 1);
                size_t upper = my_lower_bound(hits, i_start, i_end, o.ref_end + 1);
                for (size_t j = lower; j < upper; j++) {
                    if(is_added[j - i_start]) continue;
                    Hit& h = hits[j].second;
                    {
                        if (o.ref_prev_hit_startpos < h.ref_start && h.ref_start <= o.ref_end) {
                            if ((h.query_end > o.query_end) && (h.ref_end > o.ref_end)) {
                                o.query_end = h.query_end;
                                o.ref_end = h.ref_end;
                                //                        o.previous_query_start = h.query_s;
                                //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                                o.query_prev_hit_startpos = h.query_start;
                                o.ref_prev_hit_startpos = h.ref_start;
                                o.n_hits++;
                                //                        o.score += (float)1/ (float)h.count;
                                is_added[j - i_start] = true;
                                cnt_done++;
                                break;
                            } else if ((h.query_end <= o.query_end) && (h.ref_end <= o.ref_end)) {
                                //                        o.previous_query_start = h.query_s;
                                //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                                o.query_prev_hit_startpos = h.query_start;
                                o.ref_prev_hit_startpos = h.ref_start;
                                o.n_hits++;
                                //                        o.score += (float)1/ (float)h.count;
                                is_added[j - i_start] = true;
                                cnt_done++;
                                break;
                            }
                        }
                    }
                }
                if(cnt_done == i_size) break;
            }

            // Add the hit to open matches
            for(size_t j = 0; j < i_size; j++) {
                if (!is_added[j]){
                    Nam n;
                    n.query_start = hits[i_start + j].second.query_start;
                    n.query_end = hits[i_start + j].second.query_end;
                    n.ref_start = hits[i_start + j].second.ref_start;
                    n.ref_end = hits[i_start + j].second.ref_end;
                    n.ref_id = ref_id;
                    //                n.previous_query_start = h.query_s;
                    //                n.previous_ref_start = h.ref_s;
                    n.query_prev_hit_startpos = hits[i_start + j].second.query_start;
                    n.ref_prev_hit_startpos = hits[i_start + j].second.ref_start;
                    n.n_hits = 1;
                    n.is_rc = is_revcomp;
                    //                n.score += (float)1 / (float)h.count;
                    open_nams.push_back(n);
                }
            }

            // Only filter if we have advanced at least k nucleotides
            if (query_start > prev_q_start + k) {

                // Output all NAMs from open_matches to final_nams that the current hit have passed
                for (int k = 0; k < open_nams.size(); k++) {
                    Nam& n = open_nams[k];
                    if (n.query_end < query_start) {
                        int n_max_span = my_max(n.query_span(), n.ref_span());
                        int n_min_span = my_min(n.query_span(), n.ref_span());
                        float n_score;
                        n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
                        //                        n_score = n.n_hits * n.query_span();
                        n.score = n_score;
                        n.nam_id = nams.size();
                        nams.push_back(n);
                    }
                }

                // Remove all NAMs from open_matches that the current hit have passed
                auto c = query_start;
                int old_open_size = open_nams.size();
                open_nams.clear();
                for (int in = 0; in < old_open_size; ++in) {
                    if (!(open_nams[in].query_end < c)) {
                        open_nams.push_back(open_nams[in]);
                    }
                }
                prev_q_start = query_start;
            }
        }
        // Add all current open_matches to final NAMs
        for (int k = 0; k < open_nams.size(); k++) {
            Nam& n = open_nams[k];
            int n_max_span = my_max(n.query_span(), n.ref_span());
            int n_min_span = my_min(n.query_span(), n.ref_span());
            float n_score;
            n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
            //            n_score = n.n_hits * n.query_span();
            n.score = n_score;
            n.nam_id = nams.size();
            nams.push_back(n);
        }
    }
}

__device__ int lock = 0;

__device__ void acquire_lock() {
    while (atomicCAS(&lock, 0, 1) != 0) {
    }
}

__device__ void release_lock() {
    atomicExch(&lock, 0);
}

__device__ void merge_hits(
        my_vector<my_pair<int, Hit>>& hits_per_ref,
        int k,
        bool is_revcomp,
        my_vector<Nam>& nams
) {
    if(hits_per_ref.size() == 0) return;
    int num_hits = hits_per_ref.size();

    int ref_num = 0;
    my_vector<int> each_ref_size;
    int pre_ref_id = hits_per_ref[0].first;
    int now_ref_num = 1;
    for(int i = 1; i < hits_per_ref.size(); i++) {
        int ref_id = hits_per_ref[i].first;
        Hit hit = hits_per_ref[i].second;
        if(ref_id != pre_ref_id) {
            ref_num++;
            pre_ref_id = ref_id;
            each_ref_size.push_back(now_ref_num);
            now_ref_num = 1;
        } else {
            now_ref_num++;
        }
    }
    ref_num++;
    each_ref_size.push_back(now_ref_num);

    my_vector<Nam> open_nams;

    int now_vec_pos = 0;
    for (int i = 0; i < ref_num; i++) {

        if(i != 0) now_vec_pos += each_ref_size[i - 1];
        int ref_id = hits_per_ref[now_vec_pos].first;
        open_nams.clear();
        unsigned int prev_q_start = 0;

        for (int j = 0; j < each_ref_size[i]; j++) {
            Hit& h = hits_per_ref[now_vec_pos + j].second;
            bool is_added = false;
            for (int k = 0; k < open_nams.size(); k++) {
                Nam& o = open_nams[k];

                // Extend NAM
                if ((o.query_prev_hit_startpos < h.query_start) && (h.query_start <= o.query_end ) && (o.ref_prev_hit_startpos < h.ref_start) && (h.ref_start <= o.ref_end) ){
                    if ( (h.query_end > o.query_end) && (h.ref_end > o.ref_end) ) {
                        o.query_end = h.query_end;
                        o.ref_end = h.ref_end;
                        //                        o.previous_query_start = h.query_s;
                        //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                        o.query_prev_hit_startpos = h.query_start;
                        o.ref_prev_hit_startpos = h.ref_start;
                        o.n_hits ++;
                        //                        o.score += (float)1/ (float)h.count;
                        is_added = true;
                        break;
                    }
                    else if ((h.query_end <= o.query_end) && (h.ref_end <= o.ref_end)) {
                        //                        o.previous_query_start = h.query_s;
                        //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                        o.query_prev_hit_startpos = h.query_start;
                        o.ref_prev_hit_startpos = h.ref_start;
                        o.n_hits ++;
                        //                        o.score += (float)1/ (float)h.count;
                        is_added = true;
                        break;
                    }
                }

            }

            // Add the hit to open matches
            if (!is_added){
                Nam n;
                n.query_start = h.query_start;
                n.query_end = h.query_end;
                n.ref_start = h.ref_start;
                n.ref_end = h.ref_end;
                n.ref_id = ref_id;
                //                n.previous_query_start = h.query_s;
                //                n.previous_ref_start = h.ref_s;
                n.query_prev_hit_startpos = h.query_start;
                n.ref_prev_hit_startpos = h.ref_start;
                n.n_hits = 1;
                n.is_rc = is_revcomp;
                //                n.score += (float)1 / (float)h.count;
                open_nams.push_back(n);
            }

            // Only filter if we have advanced at least k nucleotides
            if (h.query_start > prev_q_start + k) {
                // Output all NAMs from open_matches to final_nams that the current hit have passed
                for (int k = 0; k < open_nams.size(); k++) {
                    Nam& n = open_nams[k];
                    if (n.query_end < h.query_start) {
                        int n_max_span = my_max(n.query_span(), n.ref_span());
                        int n_min_span = my_min(n.query_span(), n.ref_span());
                        float n_score;
                        n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
                        //                        n_score = n.n_hits * n.query_span();
                        n.score = n_score;
                        n.nam_id = nams.size();
                        nams.push_back(n);
                    }
                }

                // Remove all NAMs from open_matches that the current hit have passed
                auto c = h.query_start;
                int old_open_size = open_nams.size();
                open_nams.clear();
                for (int in = 0; in < old_open_size; ++in) {
                    if (!(open_nams[in].query_end < c)) {
                        open_nams.push_back(open_nams[in]);
                    }
                }
                prev_q_start = h.query_start;
            }
        }

        // Add all current open_matches to final NAMs
        for (int k = 0; k < open_nams.size(); k++) {
            Nam& n = open_nams[k];
            int n_max_span = my_max(n.query_span(), n.ref_span());
            int n_min_span = my_min(n.query_span(), n.ref_span());
            float n_score;
            n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
            //            n_score = n.n_hits * n.query_span();
            n.score = n_score;
            n.nam_id = nams.size();
            nams.push_back(n);
        }
    }
}

__device__ void add_to_hits_per_ref(
        my_vector<my_pair<int, Hit>>& hits_per_ref,
        int query_start,
        int query_end,
        size_t position,
        const RefRandstrobe *d_randstrobes,
        size_t d_randstrobes_size,
        int k
) {
    int min_diff = 1 << 30;
    for (const auto hash = gpu_get_hash(d_randstrobes, d_randstrobes_size, position); gpu_get_hash(d_randstrobes, d_randstrobes_size, position) == hash; ++position) {
        int ref_start = d_randstrobes[position].position;
        int ref_end = ref_start + d_randstrobes[position].strobe2_offset() + k;
        int diff = std::abs((query_end - query_start) - (ref_end - ref_start));
        if (diff <= min_diff) {
            hits_per_ref.push_back({d_randstrobes[position].reference_index(), Hit{query_start, query_end, ref_start, ref_end}});
            min_diff = diff;
        }
    }
}


#define GPU_thread_task_size 1

__global__ void gpu_rescue_get_hits(
        int bits,
        unsigned int filter_cutoff,
        int rescue_cutoff,
        const RefRandstrobe *d_randstrobes,
        size_t d_randstrobes_size,
        const my_bucket_index_t *d_randstrobe_start_indices,
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_hits_num,
        my_vector<QueryRandstrobe>* global_randstrobes,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s
)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        my_vector<my_pair<int, Hit>>* hits_per_ref0;
        my_vector<my_pair<int, Hit>>* hits_per_ref1;
        hits_per_ref0 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref1 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref0->init();
        hits_per_ref1->init();


        my_vector<RescueHit> hits_t0;
        my_vector<RescueHit> hits_t1;
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[id][i];
            //size_t position = gpu_find(d_randstrobes, d_randstrobe_start_indices, q.hash, bits);
            size_t position = q.hash;
            if (position != static_cast<size_t>(-1)) {
                if(position >= d_randstrobes_size) {
                    printf("position > d_randstrobes_size : %llu %llu\n", position, d_randstrobes_size);
                    assert(false);
                }
                unsigned int count = gpu_get_count(d_randstrobes, d_randstrobe_start_indices, position, bits);
                RescueHit rh{position, count, q.start, q.end};
                if(q.is_reverse) hits_t1.push_back(rh);
                else hits_t0.push_back(rh);
            }
        }
        global_randstrobes[id].release();
        quick_sort(&(hits_t0[0]), hits_t0.size());

        int cnt = 0;
        for (int i = 0; i < hits_t0.size(); i++) {
            RescueHit &rh = hits_t0[i];
            if ((rh.count > rescue_cutoff && cnt >= 5) || rh.count > rescue_threshold) {
                break;
            }
            add_to_hits_per_ref(*hits_per_ref0, rh.query_start, rh.query_end, rh.position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
            cnt++;
        }
        quick_sort(&(hits_t1[0]), hits_t1.size());
        cnt = 0;
        for (int i = 0; i < hits_t1.size(); i++) {
            RescueHit &rh = hits_t1[i];
            if ((rh.count > rescue_cutoff && cnt >= 5) || rh.count > rescue_threshold) {
                break;
            }
            add_to_hits_per_ref(*hits_per_ref1, rh.query_start, rh.query_end, rh.position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
            cnt++;
        }
        global_hits_num[id] = hits_per_ref0->size() + hits_per_ref1->size();
        hits_per_ref0s[id] = *hits_per_ref0;
        hits_per_ref1s[id] = *hits_per_ref1;
        my_free(hits_per_ref0);
        my_free(hits_per_ref1);
    }
}

__global__ void gpu_rescue_sort_hits(
        int num_tasks,
        IndexParameters *index_para,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s
)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;

    int l_range = bid * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;

    for (int id = l_range; id < r_range; id++) {
        sort_hits_parallel(hits_per_ref0s[id], index_para->syncmer.k, 0, tid);
        sort_hits_parallel(hits_per_ref1s[id], index_para->syncmer.k, 1, tid);
    }
}

__global__ void gpu_rescue_merge_hits(
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_nams_info,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s
)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        my_vector<Nam> nams(128);
        salign_merge_hits(hits_per_ref0s[id], index_para->syncmer.k, 0, nams);
        salign_merge_hits(hits_per_ref1s[id], index_para->syncmer.k, 1, nams);
        uint64_t local_nams_info = 0;
        for (int i = 0; i < nams.size(); i++) {
            local_nams_info += nams[i].ref_id + int(nams[i].score) + nams[i].query_start + nams[i].query_end;
        }
        global_nams_info[id] += local_nams_info;
        hits_per_ref0s[id].release();
        hits_per_ref1s[id].release();
    }
}


__global__ void gpu_get_randstrobes(
        int num_tasks,
        int *pre_sum,
        int *lens,
        char *all_seqs,
        int *pre_sum2,
        int *lens2,
        char *all_seqs2,
        IndexParameters *index_para,
        int *randstrobe_sizes,
        uint64_t *hashes,
        my_vector<QueryRandstrobe>* global_randstrobes
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    int read_num = num_tasks / 2;
    for (int id = l_range; id < r_range; id++) {
        int read_id = id % read_num;
        int rev = id / read_num;
        size_t len;
        char *seq;
        if (rev == 0) {
            len = lens[read_id];
            seq = all_seqs + pre_sum[read_id];
        } else {
            len = lens2[read_id];
            seq = all_seqs2 + pre_sum2[read_id];
        }

        my_vector<Syncmer> syncmers(len);

        const int k = index_para->syncmer.k;
        const int s = index_para->syncmer.s;
        const int t = index_para->syncmer.t_syncmer;

        const uint64_t kmask = (1ULL << 2 * k) - 1;
        const uint64_t smask = (1ULL << 2 * s) - 1;
        const uint64_t kshift = (k - 1) * 2;
        const uint64_t sshift = (s - 1) * 2;
        uint64_t gpu_qs[200];
        int l_pos = 0;
        int r_pos = 0;
        uint64_t qs_min_val = UINT64_MAX;
        int qs_min_pos = -1;
        int l = 0;
        uint64_t xk[2] = {0, 0};
        uint64_t xs[2] = {0, 0};
        for (size_t i = 0; i < len; i++) {
            int c = seq_nt4_table[(uint8_t) seq[i]];
            if (c < 4) { // not an "N" base
                xk[0] = (xk[0] << 2 | c) & kmask;                  // forward strand
                xk[1] = xk[1] >> 2 | (uint64_t)(3 - c) << kshift;  // reverse strand
                xs[0] = (xs[0] << 2 | c) & smask;                  // forward strand
                xs[1] = xs[1] >> 2 | (uint64_t)(3 - c) << sshift;  // reverse strand
                if (++l < s) {
                    continue;
                }
                // we find an s-mer
                uint64_t ys = xs[0] < xs[1] ? xs[0] : xs[1];
                uint64_t hash_s = syncmer_smer_hash(ys);
                gpu_qs[r_pos++] = hash_s;
                // not enough hashes in the queue, yet
                if (r_pos - l_pos < k - s + 1) {
                    continue;
                }
                if (r_pos - l_pos == k - s + 1) { // We are at the last s-mer within the first k-mer, need to decide if we add it
                    for (int j = l_pos; j < r_pos; j++) {
                        if (gpu_qs[j] < qs_min_val) {
                            qs_min_val = gpu_qs[j];
                            qs_min_pos = i - k + j - l_pos + 1;
                        }
                    }
                } else {
                    // update queue and current minimum and position
                    l_pos++;
                    if (qs_min_pos == i - k) { // we popped the previous minimizer, find new brute force
                        qs_min_val = UINT64_MAX;
                        qs_min_pos = i - s + 1;
                        for (int j = r_pos - 1; j >= l_pos; j--) { //Iterate in reverse to choose the rightmost minimizer in a window
                            if (gpu_qs[j] < qs_min_val) {
                                qs_min_val = gpu_qs[j];
                                qs_min_pos = i - k + j - l_pos + 1;
                            }
                        }
                    } else if (hash_s < qs_min_val) { // the new value added to queue is the new minimum
                        qs_min_val = hash_s;
                        qs_min_pos = i - s + 1;
                    }
                }
                if (qs_min_pos == i - k + t) { // occurs at t:th position in k-mer
                    uint64_t yk = xk[0] < xk[1] ? xk[0] : xk[1];
                    syncmers.push_back(Syncmer{syncmer_kmer_hash(yk), i - k + 1});
                }
            } else {
                // if there is an "N", restart
                qs_min_val = UINT64_MAX;
                qs_min_pos = -1;
                l = xs[0] = xs[1] = xk[0] = xk[1] = 0;
                r_pos = 0;
                l_pos = 0;
            }
        }


        const int w_min = index_para->randstrobe.w_min;
        const int w_max = index_para->randstrobe.w_max;
        const uint64_t q = index_para->randstrobe.q;
        const int max_dist = index_para->randstrobe.max_dist;

        my_vector<QueryRandstrobe> *randstrobes;
        randstrobes = (my_vector<QueryRandstrobe>*)my_malloc(sizeof(my_vector<QueryRandstrobe>));
        randstrobes->init((my_max(syncmers.size() - w_min, 0)) * 2);


        for (int strobe1_index = 0; strobe1_index + w_min < syncmers.size(); strobe1_index++) {
            unsigned int w_end = (strobe1_index + w_max < syncmers.size() - 1) ? (strobe1_index + w_max) : syncmers.size() - 1;
            auto strobe1 = syncmers[strobe1_index];
            auto max_position = strobe1.position + max_dist;
            unsigned int w_start = strobe1_index + w_min;
            uint64_t min_val = 0xFFFFFFFFFFFFFFFF;
            Syncmer strobe2 = strobe1;
            for (auto i = w_start; i <= w_end && syncmers[i].position <= max_position; i++) {
                uint64_t hash_diff = (strobe1.hash ^ syncmers[i].hash) & q;
                uint64_t res = __popcll(hash_diff);
                if (res < min_val) {
                    min_val = res;
                    strobe2 = syncmers[i];
                }
            }
            randstrobes->push_back(
                    QueryRandstrobe{
                            randstrobe_hash(strobe1.hash, strobe2.hash), static_cast<uint32_t>(strobe1.position),
                            static_cast<uint32_t>(strobe2.position) + index_para->syncmer.k, false
                    }
            );
        }


        for (int i = 0; i < syncmers.size() / 2; i++) {
            my_swap(syncmers[i], syncmers[syncmers.size() - i - 1]);
        }
        for (size_t i = 0; i < syncmers.size(); i++) {
            syncmers[i].position = len - syncmers[i].position - (*index_para).syncmer.k;
        }

        for (int strobe1_index = 0; strobe1_index + w_min < syncmers.size(); strobe1_index++) {
            unsigned int w_end = (strobe1_index + w_max < syncmers.size() - 1) ? (strobe1_index + w_max) : syncmers.size() - 1;
            auto strobe1 = syncmers[strobe1_index];
            auto max_position = strobe1.position + max_dist;
            unsigned int w_start = strobe1_index + w_min;
            uint64_t min_val = 0xFFFFFFFFFFFFFFFF;
            Syncmer strobe2 = strobe1;
            for (auto i = w_start; i <= w_end && syncmers[i].position <= max_position; i++) {
                uint64_t hash_diff = (strobe1.hash ^ syncmers[i].hash) & q;
                uint64_t res = __popcll(hash_diff);
                if (res < min_val) {
                    min_val = res;
                    strobe2 = syncmers[i];
                }
            }
            randstrobes->push_back(
                    QueryRandstrobe{
                            randstrobe_hash(strobe1.hash, strobe2.hash), static_cast<uint32_t>(strobe1.position),
                            static_cast<uint32_t>(strobe2.position) + index_para->syncmer.k, true
                    }
            );
        }


        randstrobe_sizes[id] += randstrobes->size();
        for (int i = 0; i < randstrobes->size(); i++) hashes[id] += (*randstrobes)[i].hash;
        global_randstrobes[id] = *randstrobes;
        my_free(randstrobes);
//        randstrobe_sizes[id] += syncmers.size();
    }
}

__global__ void gpu_get_hits_after(
        int bits,
        unsigned int filter_cutoff,
        int rescue_cutoff,
        const RefRandstrobe *d_randstrobes,
        size_t d_randstrobes_size,
        const my_bucket_index_t *d_randstrobe_start_indices,
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_hits_num,
        my_vector<QueryRandstrobe>* global_randstrobes,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {

        int sum_seeds0 = 0;
        int sum_seeds1 = 0;
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
//            size_t position = global_randstrobes[id][i].hash;
//            if (position == static_cast<size_t>(-1)) continue;
//            bool res = gpu_is_filtered(d_randstrobes, d_randstrobes_size, position, filter_cutoff);
//            if (res) continue;
            if (global_randstrobes[id][i].is_reverse) {
                sum_seeds1++;
            } else {
                sum_seeds0++;
            }
        }
        my_vector<my_pair<int, Hit>>* hits_per_ref0;
        my_vector<my_pair<int, Hit>>* hits_per_ref1;
        hits_per_ref0 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref1 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref0->init(sum_seeds0 * 2);
        hits_per_ref1->init(sum_seeds1 * 2);

        uint64_t local_total_hits = 0;
        uint64_t local_nr_good_hits = 0;
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[id][i];
            size_t position = q.hash;
            if (position != static_cast<size_t>(-1)) {
                local_total_hits++;
                bool res = gpu_is_filtered(d_randstrobes, d_randstrobes_size, position, filter_cutoff);
                if (res) continue;
                local_nr_good_hits++;
                if(q.is_reverse) {
                    add_to_hits_per_ref(*hits_per_ref1, q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                } else {
                    add_to_hits_per_ref(*hits_per_ref0, q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                }
            }
        }
        float nonrepetitive_fraction = local_total_hits > 0 ? ((float) local_nr_good_hits) / ((float) local_total_hits) : 1.0;

        if (nonrepetitive_fraction < 0.7 || hits_per_ref0->size() + hits_per_ref1->size() == 0) {
        } else {
            global_randstrobes[id].release();
        }
        global_hits_num[id] = hits_per_ref0->size() + hits_per_ref1->size();
        hits_per_ref0s[id] = *hits_per_ref0;
        hits_per_ref1s[id] = *hits_per_ref1;
        my_free(hits_per_ref0);
        my_free(hits_per_ref1);
    }
}

__global__ void gpu_get_hits_pre(
        int bits,
        unsigned int filter_cutoff,
        int rescue_cutoff,
        const RefRandstrobe *d_randstrobes,
        size_t d_randstrobes_size,
        const my_bucket_index_t *d_randstrobe_start_indices,
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_hits_num,
        my_vector<QueryRandstrobe>* global_randstrobes,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[id][i];
            size_t position = gpu_find(d_randstrobes, d_randstrobe_start_indices, q.hash, bits);
            //if(position != static_cast<size_t>(-1) && position >= d_randstrobes_size) {
            //    printf("position GG %zu %zu\n", position, d_randstrobes_size);
            //    assert(false);
            //}
            global_randstrobes[id][i].hash = position;
        }
    }
}

__global__ void gpu_get_hits(
        int bits,
        unsigned int filter_cutoff,
        int rescue_cutoff,
        const RefRandstrobe *d_randstrobes,
        size_t d_randstrobes_size,
        const my_bucket_index_t *d_randstrobe_start_indices,
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_hits_num,
        my_vector<QueryRandstrobe>* global_randstrobes,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int read_id = id / 2;
        int rev = id % 2;

        my_vector<my_pair<int, Hit>>* hits_per_ref0;
        my_vector<my_pair<int, Hit>>* hits_per_ref1;
        hits_per_ref0 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref1 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref0->init();
        hits_per_ref1->init();

        uint64_t local_total_hits = 0;
        uint64_t local_nr_good_hits = 0;
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[id][i];
            size_t position = gpu_find(d_randstrobes, d_randstrobe_start_indices, q.hash, bits);
            global_randstrobes[id][i].hash = position;
            if (position != static_cast<size_t>(-1)) {
                local_total_hits++;
                bool res = gpu_is_filtered(d_randstrobes, d_randstrobes_size, position, filter_cutoff);
                if (res) continue;
                local_nr_good_hits++;
                if(q.is_reverse) {
                    add_to_hits_per_ref(*hits_per_ref1, q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                } else {
                    add_to_hits_per_ref(*hits_per_ref0, q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                }
            }
        }
        float nonrepetitive_fraction = local_total_hits > 0 ? ((float) local_nr_good_hits) / ((float) local_total_hits) : 1.0;

        if (nonrepetitive_fraction < 0.7 || hits_per_ref0->size() + hits_per_ref1->size() == 0) {

        } else {
            global_randstrobes[id].release();
        }
        global_hits_num[id] = hits_per_ref0->size() + hits_per_ref1->size();
        hits_per_ref0s[id] = *hits_per_ref0;
        hits_per_ref1s[id] = *hits_per_ref1;
        my_free(hits_per_ref0);
        my_free(hits_per_ref1);
    }
}

__global__ void gpu_get_randstrobes_and_hits(
        int bits,
        unsigned int filter_cutoff,
        int rescue_cutoff,
        const RefRandstrobe *d_randstrobes,
        size_t d_randstrobes_size,
        const my_bucket_index_t *d_randstrobe_start_indices,
        IndexParameters *index_para,
        int num_tasks,
        int *pre_sum,
        int *lens,
        char *all_seqs,
        int *pre_sum2,
        int *lens2,
        char *all_seqs2,
        uint64_t *global_hits_num,
        my_vector<QueryRandstrobe>* global_randstrobes,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s,
        int *randstrobe_sizes,
        uint64_t *hashes
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int read_id = id / 2;
        int rev = id % 2;
        size_t len;
        char *seq;
        if (rev == 0) {
            len = lens[read_id];
            seq = all_seqs + pre_sum[read_id];
        } else {
            len = lens2[read_id];
            seq = all_seqs2 + pre_sum2[read_id];
        }

        my_vector<QueryRandstrobe> *randstrobes;
        randstrobes = (my_vector<QueryRandstrobe>*)my_malloc(sizeof(my_vector<QueryRandstrobe>));
        randstrobes->init();

        my_vector<Syncmer> syncmers;
        my_vector<uint64_t> vec4syncmers;

        SyncmerIterator syncmer_iterator{&vec4syncmers, seq, len, (*index_para).syncmer};
        Syncmer syncmer;
        while (1) {
            syncmer = syncmer_iterator.gpu_next();
            if (syncmer.is_end()) break;
            syncmers.push_back(syncmer);
        }
        if (syncmers.size() != 0)  {
            RandstrobeIterator randstrobe_fwd_iter{&syncmers, (*index_para).randstrobe};
            while (randstrobe_fwd_iter.gpu_has_next()) {
                Randstrobe randstrobe = randstrobe_fwd_iter.gpu_next();
                randstrobes->push_back(
                        QueryRandstrobe{
                                randstrobe.hash, randstrobe.strobe1_pos,
                                randstrobe.strobe2_pos + (*index_para).syncmer.k, false
                        }
                );
            }
            for (int i = 0; i < syncmers.size() / 2; i++) {
                my_swap(syncmers[i], syncmers[syncmers.size() - i - 1]);
            }
            for (size_t i = 0; i < syncmers.size(); i++) {
                syncmers[i].position = len - syncmers[i].position - (*index_para).syncmer.k;
            }
            RandstrobeIterator randstrobe_rc_iter{&syncmers, (*index_para).randstrobe};
            while (randstrobe_rc_iter.gpu_has_next()) {
                Randstrobe randstrobe = randstrobe_rc_iter.gpu_next();
                randstrobes->push_back(
                        QueryRandstrobe{
                                randstrobe.hash, randstrobe.strobe1_pos,
                                randstrobe.strobe2_pos + (*index_para).syncmer.k, true
                        }
                );
            }
        }

        randstrobe_sizes[id] += randstrobes->size();
        for (int i = 0; i < randstrobes->size(); i++) hashes[id] += (*randstrobes)[i].hash;

        my_vector<my_pair<int, Hit>>* hits_per_ref0;
        my_vector<my_pair<int, Hit>>* hits_per_ref1;
        hits_per_ref0 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref1 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref0->init();
        hits_per_ref1->init();

        uint64_t local_total_hits = 0;
        uint64_t local_nr_good_hits = 0;
        for (int i = 0; i < randstrobes->size(); i++) {
            QueryRandstrobe q = (*randstrobes)[i];
            size_t position = gpu_find(d_randstrobes, d_randstrobe_start_indices, q.hash, bits);
            if (position != static_cast<size_t>(-1)) {
                local_total_hits++;
                bool res = gpu_is_filtered(d_randstrobes, d_randstrobes_size, position, filter_cutoff);
                if (res) continue;
                local_nr_good_hits++;
                if(q.is_reverse) {
                    add_to_hits_per_ref(*hits_per_ref1, q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                } else {
                    add_to_hits_per_ref(*hits_per_ref0, q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                }
            }
        }
        float nonrepetitive_fraction = local_total_hits > 0 ? ((float) local_nr_good_hits) / ((float) local_total_hits) : 1.0;
        if (nonrepetitive_fraction < 0.7 || hits_per_ref0->size() + hits_per_ref1->size() == 0) {
            global_randstrobes[id] = *randstrobes;
        } else {
            randstrobes->release();
        }
        global_hits_num[id] = hits_per_ref0->size() + hits_per_ref1->size();
        hits_per_ref0s[id] = *hits_per_ref0;
        hits_per_ref1s[id] = *hits_per_ref1;
        my_free(hits_per_ref0);
        my_free(hits_per_ref1);
        my_free(randstrobes);
    }
}

__global__ void gpu_sort_hits(
        int num_tasks,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        sort_hits_single(hits_per_ref0s[id]);
        sort_hits_single(hits_per_ref1s[id]);
        //sort_hits_by_refid(hits_per_ref0s[id]);
        //sort_hits_by_refid(hits_per_ref1s[id]);
    }
}

__global__ void gpu_merge_hits(
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_nams_info,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        my_vector<Nam> nams;
        salign_merge_hits(hits_per_ref0s[id], index_para->syncmer.k, 0, nams);
        salign_merge_hits(hits_per_ref1s[id], index_para->syncmer.k, 1, nams);
        uint64_t local_nams_info = 0;
        for (int i = 0; i < nams.size(); i++) {
            local_nams_info += nams[i].ref_id + int(nams[i].score) + nams[i].query_start + nams[i].query_end;
        }
        global_nams_info[id] += local_nams_info;
        hits_per_ref0s[id].release();
        hits_per_ref1s[id].release();
    }
}

__global__ void gpu_align_PE(
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_align_info,
        AlignmentParameters* aligner_parameters,
        int *pre_sum,
        int *lens,
        char *all_seqs,
        int *pre_sum2,
        int *lens2,
        char *all_seqs2,
        GPUReferences *global_references,
        MappingParameters *mapping_parameters,
        my_vector<Nam> *global_nams
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        size_t seq_len1, seq_len2;
        char *seq1, *seq2;
        seq_len1 = lens[id];
        seq1 = all_seqs + pre_sum[id];
        seq_len2 = lens2[id];
        seq2 = all_seqs2 + pre_sum2[id];

        GPUAlignTmpRes align_tmp_res;
        align_tmp_res.init();
        GPUInsertSizeDistribution isize_est;
        align_PE_part(align_tmp_res, *aligner_parameters, global_nams[id], global_nams[id + num_tasks], seq1, seq_len1, seq2, seq_len2, index_para->syncmer.k, *global_references,
                      mapping_parameters->dropoff_threshold, isize_est, mapping_parameters->max_tries, mapping_parameters->max_secondary);
        global_align_info[id] += align_tmp_res.type + align_tmp_res.mapq1 + align_tmp_res.mapq2 + align_tmp_res.type4_loop_size;
        global_align_info[id] += align_tmp_res.is_extend_seed.size() + align_tmp_res.consistent_nam.size() + align_tmp_res.is_read1.size() +
                align_tmp_res.type4_nams.size() + align_tmp_res.todo_nams.size() + align_tmp_res.done_align.size() + align_tmp_res.align_res.size();
        uint64_t local_sum = 0;
        for (int i = 0; i < align_tmp_res.todo_nams.size(); i++) {
            local_sum += align_tmp_res.todo_nams[i].ref_id;
        }
        global_align_info[id] += local_sum;
        global_nams[id].release();
        global_nams[id + num_tasks].release();
    }
}

__global__ void gpu_merge_hits_get_nams(
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_nams_info,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s,
        my_vector<Nam> *global_nams
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int real_id = id;
        my_vector<Nam> *nams1 = (my_vector<Nam>*)my_malloc(sizeof(my_vector<Nam>));
        nams1->init();
        salign_merge_hits(hits_per_ref0s[real_id], index_para->syncmer.k, 0, *nams1);
        salign_merge_hits(hits_per_ref1s[real_id], index_para->syncmer.k, 1, *nams1);
        uint64_t local_nams_info = 0;
        for (int i = 0; i < nams1->size(); i++) {
            local_nams_info += (*nams1)[i].ref_id + int((*nams1)[i].score) + (*nams1)[i].query_start + (*nams1)[i].query_end;
        }
        global_nams_info[real_id] += local_nams_info;
        global_nams[real_id] = *nams1;
        my_free(nams1);
        hits_per_ref0s[real_id].release();
        hits_per_ref1s[real_id].release();

        real_id = id + num_tasks;
        my_vector<Nam> *nams2 = (my_vector<Nam>*)my_malloc(sizeof(my_vector<Nam>));
        nams2->init();
        salign_merge_hits(hits_per_ref0s[real_id], index_para->syncmer.k, 0, *nams2);
        salign_merge_hits(hits_per_ref1s[real_id], index_para->syncmer.k, 1, *nams2);
        local_nams_info = 0;
        for (int i = 0; i < nams2->size(); i++) {
            local_nams_info += (*nams2)[i].ref_id + int((*nams2)[i].score) + (*nams2)[i].query_start + (*nams2)[i].query_end;
        }
        global_nams_info[real_id] += local_nams_info;
        global_nams[real_id] = *nams2;
        my_free(nams2);
        hits_per_ref0s[real_id].release();
        hits_per_ref1s[real_id].release();
    }
}


klibpp::KSeq ConvertNeo2KSeq(neoReference ref) {
    klibpp::KSeq res;
    res.name = std::string((char *) ref.base + ref.pname, ref.lname);
    if (!res.name.empty()) {
        size_t space_pos = res.name.find(' ');
        int l_pos = 0;
        if (res.name[0] == '@') l_pos = 1;
        if (space_pos != std::string::npos) {
            res.name = res.name.substr(l_pos, space_pos - l_pos);
        } else {
            res.name = res.name.substr(l_pos);
        }
    }
    res.seq = std::string((char *) ref.base + ref.pseq, ref.lseq);
    res.comment = std::string((char *) ref.base + ref.pstrand, ref.lstrand);
    res.qual = std::string((char *) ref.base + ref.pqual, ref.lqual);
    return res;
}

static Logger &logger = Logger::get();


int main(int argc, char **argv) {
    auto opt = parse_command_line_arguments(argc, argv);

    InputBuffer input_buffer = get_input_buffer(opt);
    if (!opt.r_set && !opt.reads_filename1.empty()) {
        opt.r = estimate_read_length(input_buffer);
        logger.info() << "Estimated read length: " << opt.r << " bp\n";

    }

    IndexParameters index_parameters = IndexParameters::from_read_length(
            opt.r,
            opt.k_set ? opt.k : IndexParameters::DEFAULT,
            opt.s_set ? opt.s : IndexParameters::DEFAULT,
            opt.l_set ? opt.l : IndexParameters::DEFAULT,
            opt.u_set ? opt.u : IndexParameters::DEFAULT,
            opt.c_set ? opt.c : IndexParameters::DEFAULT,
            opt.max_seed_len_set ? opt.max_seed_len : IndexParameters::DEFAULT
    );
    AlignmentParameters aln_params;
    aln_params.match = opt.A;
    aln_params.mismatch = opt.B;
    aln_params.gap_open = opt.O;
    aln_params.gap_extend = opt.E;
    aln_params.end_bonus = opt.end_bonus;

    AlignmentParameters *d_aligner;
    hipMalloc(&d_aligner, sizeof(AlignmentParameters));
    hipMemcpy(d_aligner, &aln_params, sizeof(AlignmentParameters), hipMemcpyHostToDevice);


    MappingParameters map_param;
    map_param.r = opt.r;
    map_param.max_secondary = opt.max_secondary;
    map_param.dropoff_threshold = opt.dropoff_threshold;
    map_param.rescue_level = opt.rescue_level;
    map_param.max_tries = opt.max_tries;
    map_param.is_sam_out = opt.is_sam_out;
    map_param.cigar_ops = opt.cigar_eqx ? CigarOps::EQX : CigarOps::M;
    map_param.output_unmapped = opt.output_unmapped;
    map_param.details = opt.details;
    map_param.verify();

    MappingParameters* d_map_param;
    hipMalloc(&d_map_param, sizeof(MappingParameters));
    hipMemcpy(d_map_param, &map_param, sizeof(MappingParameters), hipMemcpyHostToDevice);

    std::string index_file_path = argv[1];
    References references;
    references = References::from_fasta(opt.ref_filename);
    logger.info() << "Reference size: " << references.total_length() / 1E6 << " Mbp ("
                  << references.size() << " contig" << (references.size() == 1 ? "" : "s")
                  << "; largest: "
                  << (*std::max_element(references.lengths.begin(), references.lengths.end()) / 1E6) << " Mbp)\n";
    if (references.total_length() == 0) {
        throw InvalidFasta("No reference sequences found");
    }
    StrobemerIndex index(references, index_parameters, opt.bits);
    std::string sti_path = opt.ref_filename + index_parameters.filename_extension();
    index.read(sti_path);

    int para_rescue_cutoff = opt.rescue_level < 100 ? opt.rescue_level * index.filter_cutoff : rescue_threshold;

    std::cout << "rescue_cutoff: " << para_rescue_cutoff << std::endl;
    std::cout << "filter_cutoff: " << index.filter_cutoff << std::endl;
    std::cout << "rescue_level: " << opt.rescue_level << std::endl;

    std::cout << "read file : " << opt.reads_filename1 << " " << opt.reads_filename2 << std::endl;

    rabbit::fq::FastqDataPool fastqPool(1024, 1 << 22);
    rabbit::core::TDataQueue<rabbit::fq::FastqDataPairChunk> queue_pe(1024, 1);
    std::thread *producer;
    producer = new std::thread(producer_pe_fastq_task, opt.reads_filename1, opt.reads_filename2, std::ref(fastqPool),
                               std::ref(queue_pe));

    std::vector<neoReference> data1;
    std::vector<neoReference> data2;
    rabbit::fq::FastqDataPairChunk *fqdatachunk = new rabbit::fq::FastqDataPairChunk;
    std::vector<klibpp::KSeq> records1;
    std::vector<klibpp::KSeq> records2;
    long long id;
    while (queue_pe.Pop(id, fqdatachunk)) {
        data1.clear();
        data2.clear();
        rabbit::fq::chunkFormat((rabbit::fq::FastqDataChunk *) (fqdatachunk->left_part), data1);
        rabbit::fq::chunkFormat((rabbit::fq::FastqDataChunk *) (fqdatachunk->right_part), data2);
        assert(data1.size() == data2.size());
        for (int i = 0; i < data1.size(); i++) {
            auto item1 = data1[i];
            auto item2 = data2[i];
            records1.push_back(ConvertNeo2KSeq(item1));
            records2.push_back(ConvertNeo2KSeq(item2));
        }
        fastqPool.Release(fqdatachunk->left_part);
        fastqPool.Release(fqdatachunk->right_part);
    }
    producer->join();
    printf("read file done, %zu %zu\n", records1.size(), records2.size());

    double t0;

    t0 = GetTime();
    GPUReferences *global_references;
    hipMallocManaged(&global_references, sizeof(GPUReferences));
    global_references->num_refs = references.size();
    hipMalloc(&global_references->sequences.data, references.size() * sizeof(my_string));
    global_references->sequences.length = references.size();
    global_references->sequences.capacity = references.size();
    for (int i = 0; i < references.size(); i++) {
        my_string ref;
        ref.slen = references.lengths[i];
        hipMalloc(&ref.data, references.lengths[i]);
        hipMemcpy(ref.data, references.sequences[i].data(), references.lengths[i], hipMemcpyHostToDevice);
        hipMemcpy(global_references->sequences.data + i, &ref, sizeof(my_string), hipMemcpyHostToDevice);
    }
    hipMalloc(&global_references->lengths.data, references.size() * sizeof(int));
    hipMemcpy(global_references->lengths.data, references.lengths.data(), references.size() * sizeof(int), hipMemcpyHostToDevice);
    global_references->lengths.length = references.size();
    global_references->lengths.capacity = references.size();

    RefRandstrobe *d_randstrobes;
    my_bucket_index_t *d_randstrobe_start_indices;
    std::cout << index.randstrobes.size() * sizeof(RefRandstrobe) << std::endl;
    hipMalloc(&d_randstrobes, index.randstrobes.size() * sizeof(RefRandstrobe));
    hipMalloc(&d_randstrobe_start_indices, index.randstrobe_start_indices.size() * sizeof(my_bucket_index_t));
    hipMemset(d_randstrobes, 0, index.randstrobes.size() * sizeof(RefRandstrobe));
    hipMemset(d_randstrobe_start_indices, 0, index.randstrobe_start_indices.size() * sizeof(my_bucket_index_t));
    std::cout << "malloc1 execution time: " << GetTime() - t0 << " seconds, size "
              << index.randstrobes.size() * sizeof(RefRandstrobe) +
                 index.randstrobe_start_indices.size() * sizeof(my_bucket_index_t) << std::endl;

    t0 = GetTime();
    hipMemcpy(d_randstrobes, index.randstrobes.data(), index.randstrobes.size() * sizeof(RefRandstrobe),
               hipMemcpyHostToDevice);
    hipMemcpy(d_randstrobe_start_indices, index.randstrobe_start_indices.data(),
               index.randstrobe_start_indices.size() * sizeof(my_bucket_index_t), hipMemcpyHostToDevice);
    std::cout << "memcpy1 execution time: " << GetTime() - t0 << " seconds, size "
              << index.randstrobes.size() * sizeof(RefRandstrobe) +
                 index.randstrobe_start_indices.size() * sizeof(my_bucket_index_t) << std::endl;


#define batch_size 400000ll
#define batch_seq_szie batch_size * 250ll


    uint64_t num_bytes = 24ll * 1024ll * 1024ll * 1024ll;
    uint64_t seed = 13;
    init_mm(num_bytes, seed);

    printf("Gallatin global allocator initialized with %lu bytes.\n", num_bytes);

    my_vector<QueryRandstrobe> *global_randstrobes;
    hipMallocManaged(&global_randstrobes, batch_size * 2 * sizeof(my_vector<QueryRandstrobe>));

    int *global_sorted_ids;
    hipMallocManaged(&global_sorted_ids, batch_size * 4 * sizeof(int));

    int *global_randstrobe_sizes;
    hipMallocManaged(&global_randstrobe_sizes, batch_size * 2 * sizeof(int));
    uint64_t * global_hashes_value;
    hipMallocManaged(&global_hashes_value, batch_size * 2 * sizeof(uint64_t));

    my_vector<my_pair<int, Hit>> *global_hits_per_ref0s;
    hipMallocManaged(&global_hits_per_ref0s, batch_size * 2 * sizeof(my_vector<my_pair<int, Hit>>));

    my_vector<my_pair<int, Hit>> *global_hits_per_ref1s;
    hipMallocManaged(&global_hits_per_ref1s, batch_size * 2 * sizeof(my_vector<my_pair<int, Hit>>));

    my_vector<Nam> *global_nams;
    hipMallocManaged(&global_nams, batch_size * 2 * sizeof(my_vector<Nam>));


    t0 = GetTime();
    char *d_seq;
    int *d_len;
    int *d_pre_sum;
    hipMalloc(&d_seq, batch_seq_szie);
    hipMemset(d_seq, 0, batch_seq_szie);
    hipMalloc(&d_len, batch_size * sizeof(int));
    hipMemset(d_len, 0, batch_size * sizeof(int));
    hipMalloc(&d_pre_sum, batch_size * sizeof(int));
    hipMemset(d_pre_sum, 0, batch_size * sizeof(int));

    char *d_seq2;
    int *d_len2;
    int *d_pre_sum2;
    hipMalloc(&d_seq2, batch_seq_szie);
    hipMemset(d_seq2, 0, batch_seq_szie);
    hipMalloc(&d_len2, batch_size * sizeof(int));
    hipMemset(d_len2, 0, batch_size * sizeof(int));
    hipMalloc(&d_pre_sum2, batch_size * sizeof(int));
    hipMemset(d_pre_sum2, 0, batch_size * sizeof(int));
    std::cout << "malloc2 execution time: " << GetTime() - t0 << " seconds, size "
              << batch_seq_szie + batch_size * sizeof(int) << std::endl;

    IndexParameters *d_index_para;
    hipMalloc(&d_index_para, sizeof(IndexParameters));


    int *h_len = new int[batch_size];
    int *h_pre_sum = new int[batch_size + 1];
    char *h_seq = new char[batch_seq_szie];

    int *h_len2 = new int[batch_size];
    int *h_pre_sum2 = new int[batch_size + 1];
    char *h_seq2 = new char[batch_seq_szie];

    double gpu_cost1 = 0;
    double gpu_cost2 = 0;
    double gpu_cost2_1 = 0;
    double gpu_cost2_2 = 0;
    double gpu_cost3 = 0;
    double gpu_cost4 = 0;
    double gpu_cost5 = 0;
    double gpu_cost6 = 0;
    double gpu_cost7 = 0;
    double gpu_cost8 = 0;
    double cpu_sort = 0;
    double tot_cost = 0;

    uint64_t check_sum = 0;
    uint64_t size_tot = 0;

    uint64_t global_hits_num12 = 0;
    uint64_t global_hits_num3 = 0;

    uint64_t * global_hits_num;
    hipMallocManaged(&global_hits_num, batch_size * 2 * sizeof(uint64_t));

    uint64_t global_nams_info12 = 0;
    uint64_t global_nams_info3 = 0;
    uint64_t * global_nams_info;
    hipMallocManaged(&global_nams_info, batch_size * 2 * sizeof(uint64_t));

    uint64_t global_align_info12 = 0;
    uint64_t global_align_info3 = 0;
    uint64_t * global_align_info;
    hipMallocManaged(&global_align_info, batch_size * sizeof(uint64_t));



    assert(records1.size() == records2.size());

//    print_mm();

    t0 = GetTime();
    for (int l_id = 0; l_id < records1.size(); l_id += batch_size) {
        printf("process %d / %d\n", l_id, records1.size());
        int r_id = l_id + batch_size;
        if (r_id > records1.size()) r_id = records1.size();
        int s_len = r_id - l_id;

        double t1;

        uint64_t tot_len = 0;
        uint64_t tot_len2 = 0;
        h_pre_sum[0] = 0;
        h_pre_sum2[0] = 0;
        for (int i = l_id; i < r_id; i++) {
            tot_len += records1[i].seq.length();
            tot_len2 += records2[i].seq.length();
            h_len[i - l_id] = records1[i].seq.length();
            h_len2[i - l_id] = records2[i].seq.length();
            h_pre_sum[i + 1 - l_id] = h_pre_sum[i - l_id] + h_len[i - l_id];
            h_pre_sum2[i + 1 - l_id] = h_pre_sum2[i - l_id] + h_len2[i - l_id];
        }
#pragma omp parallel for
        for (int i = l_id; i < r_id; i++) {
            memcpy(h_seq + h_pre_sum[i - l_id], records1[i].seq.c_str(), h_len[i - l_id]);
            memcpy(h_seq2 + h_pre_sum2[i - l_id], records2[i].seq.c_str(), h_len2[i - l_id]);
        }

        hipMemcpy(d_seq, h_seq, tot_len, hipMemcpyHostToDevice);
        hipMemcpy(d_len, h_len, s_len * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_pre_sum, h_pre_sum, s_len * sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(d_seq2, h_seq2, tot_len2, hipMemcpyHostToDevice);
        hipMemcpy(d_len2, h_len2, s_len * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_pre_sum2, h_pre_sum2, s_len * sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(d_index_para, &index_parameters, sizeof(IndexParameters), hipMemcpyHostToDevice);

        for (int i = 0; i < s_len; i++) {
            global_align_info[i] = 0;
        }

        for (int i = 0; i < s_len * 2; i++) {
            // check infos
            global_randstrobe_sizes[i] = 0;
            global_hashes_value[i] = 0;
            global_hits_num[i] = 0;
            global_nams_info[i] = 0;

            global_hits_per_ref0s[i].data = nullptr;
            global_hits_per_ref0s[i].length = 0;
            global_hits_per_ref1s[i].data = nullptr;
            global_hits_per_ref1s[i].length = 0;

            global_randstrobes[i].data = nullptr;
            global_randstrobes[i].length = 0;

            global_nams[i].data = nullptr;
            global_nams[i].length = 0;
        }


        t1 = GetTime();
        int threads_per_block;
        int reads_per_block;
        int blocks_per_grid;

        threads_per_block = 1;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (s_len * 2 + reads_per_block - 1) / reads_per_block;
        gpu_get_randstrobes<<<blocks_per_grid, threads_per_block>>>(s_len * 2, d_pre_sum, d_len, d_seq, d_pre_sum2, d_len2, d_seq2, d_index_para,
                                                                    global_randstrobe_sizes, global_hashes_value, global_randstrobes);
        hipDeviceSynchronize();
        gpu_cost1 += GetTime() - t1;

        t1 = GetTime();

        double t11 = GetTime();
        threads_per_block = 1;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (s_len * 2 + reads_per_block - 1) / reads_per_block;
        gpu_get_hits_pre<<<blocks_per_grid, threads_per_block>>>(index.bits, index.filter_cutoff, para_rescue_cutoff, d_randstrobes, index.randstrobes.size(), d_randstrobe_start_indices,
                                                             s_len * 2, d_index_para, global_hits_num, global_randstrobes,
                                                             global_hits_per_ref0s, global_hits_per_ref1s);
        hipDeviceSynchronize();
        gpu_cost2_1 += GetTime() - t11;

      
        t11 = GetTime();
        threads_per_block = 1;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (s_len * 2 + reads_per_block - 1) / reads_per_block;
        gpu_get_hits_after<<<blocks_per_grid, threads_per_block>>>(index.bits, index.filter_cutoff, para_rescue_cutoff, d_randstrobes, index.randstrobes.size(), d_randstrobe_start_indices,
                                                             s_len * 2, d_index_para, global_hits_num, global_randstrobes,
                                                             global_hits_per_ref0s, global_hits_per_ref1s);
        hipDeviceSynchronize();
        gpu_cost2_2 += GetTime() - t11;

        gpu_cost2 += GetTime() - t1;


        t1 = GetTime();
        threads_per_block = 1;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (s_len * 2 + reads_per_block - 1) / reads_per_block;
        gpu_sort_hits<<<blocks_per_grid, threads_per_block>>>(s_len * 2, global_hits_per_ref0s, global_hits_per_ref1s);
        hipDeviceSynchronize();
        gpu_cost3 += GetTime() - t1;

        t1 = GetTime();
        threads_per_block = 1;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (s_len + reads_per_block - 1) / reads_per_block;
        gpu_merge_hits_get_nams<<<blocks_per_grid, threads_per_block>>>(s_len, d_index_para, global_nams_info, global_hits_per_ref0s, global_hits_per_ref1s, global_nams);
        hipDeviceSynchronize();
        gpu_cost4 += GetTime() - t1;
        printf("get nams done\n");

        t1 = GetTime();
        threads_per_block = 1;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (s_len + reads_per_block - 1) / reads_per_block;
        gpu_align_PE<<<blocks_per_grid, threads_per_block>>>(s_len, d_index_para, global_align_info, d_aligner, d_pre_sum, d_len, d_seq, d_pre_sum2, d_len2, d_seq2,
                                                             global_references, d_map_param, global_nams);
        hipDeviceSynchronize();
        gpu_cost5 += GetTime() - t1;


        for (size_t i = 0; i < s_len * 2; ++i) {
            size_tot += global_randstrobe_sizes[i];
            check_sum += global_hashes_value[i];
            global_hits_num12 += global_hits_num[i];
            global_nams_info12 += global_nams_info[i];
        }

        for (int i = 0; i < s_len; i++) {
            global_align_info12 += global_align_info[i];
        }

		int rescue_num = 0;
        for(int i = 0; i < s_len * 2; i++) {
            if (global_randstrobes[i].data != nullptr) {
                global_randstrobes[rescue_num] = global_randstrobes[i];
                rescue_num++;
            }
        }

        printf("rescue read num %d\n", rescue_num);

        for (int i = 0; i < rescue_num; i++) {
            global_hits_num[i] = 0;
            global_nams_info[i] = 0;

            global_hits_per_ref0s[i].data = nullptr;
            global_hits_per_ref0s[i].length = 0;
            global_hits_per_ref1s[i].data = nullptr;
            global_hits_per_ref1s[i].length = 0;
        }

        t1 = GetTime();
        threads_per_block = 1;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (rescue_num + reads_per_block - 1) / reads_per_block;
        gpu_rescue_get_hits<<<blocks_per_grid, threads_per_block>>>(index.bits, index.filter_cutoff, para_rescue_cutoff, d_randstrobes, index.randstrobes.size(), d_randstrobe_start_indices,
                                                             rescue_num, d_index_para, global_hits_num, global_randstrobes,
                                                             global_hits_per_ref0s, global_hits_per_ref1s);
        hipDeviceSynchronize();
        gpu_cost6 += GetTime() - t1;

        t1 = GetTime();
        threads_per_block = 1;
        reads_per_block = GPU_thread_task_size;
        blocks_per_grid = (rescue_num + reads_per_block - 1) / reads_per_block;
        //gpu_rescue_sort_hits<<<rescue_num, threads_per_block>>>(rescue_num, d_index_para, global_hits_per_ref0s, global_hits_per_ref1s);
        gpu_sort_hits<<<blocks_per_grid, threads_per_block>>>(rescue_num, global_hits_per_ref0s, global_hits_per_ref1s);
        hipDeviceSynchronize();
        gpu_cost7 += GetTime() - t1;



        t1 = GetTime();
        threads_per_block = 1;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (rescue_num + reads_per_block - 1) / reads_per_block;
        gpu_rescue_merge_hits<<<blocks_per_grid, threads_per_block>>>(rescue_num, d_index_para, global_nams_info, global_hits_per_ref0s, global_hits_per_ref1s);
        hipDeviceSynchronize();
        gpu_cost8 += GetTime() - t1;

        for (int i = 0; i < rescue_num; ++i) {
            global_hits_num3 += global_hits_num[i];
            global_nams_info3 += global_nams_info[i];
        }

//        print_mm();

    }
    tot_cost += GetTime() - t0;

    std::cout << "gpu cost " << gpu_cost1 << " " << gpu_cost2 << " [" << gpu_cost2_1 << " " << gpu_cost2_2 << "] " << gpu_cost3 << " " << gpu_cost4 << " " << gpu_cost5 << std::endl;
    std::cout << gpu_cost6 << " " << gpu_cost7 << " " << gpu_cost8 << std::endl;
    std::cout << "cpu sort cost " << cpu_sort << std::endl;
    std::cout << "total cost " << tot_cost << std::endl;
    std::cout << "check_sum : " << check_sum << ", size_tot : " << size_tot << std::endl;
    std::cout << "total_hits12 : " << global_hits_num12 << ", nr_good_hits12 : " << global_nams_info12 << std::endl;
    std::cout << "total_hits3 : " << global_hits_num3 << ", nr_good_hits3 : " << global_nams_info3 << std::endl;
    std::cout << "total_align_info12 : " << global_align_info12 << ", nr_good_align_info3 : " << global_align_info3 << std::endl;

    t0 = GetTime();
    hipFree(d_seq);
    hipFree(d_len);
    hipFree(d_pre_sum);
    hipFree(d_index_para);
    hipFree(d_randstrobes);
    hipFree(d_randstrobe_start_indices);
    delete h_seq;
    delete h_len;
    delete h_pre_sum;
    std::cout << "free execution time: " << GetTime() - t0 << " seconds" << std::endl;

    free_mm();
    return 0;
}

