#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <stdexcept>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstring> // For strerror
#include <sys/time.h>
#include <thread>
#include <omp.h>
#include <unistd.h>

inline double GetTime() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_sec + (double) tv.tv_usec / 1000000;
}


using randstrobe_hash_t = uint64_t;
using bucket_index_t = size_t;

struct RefRandstrobe {
    randstrobe_hash_t hash;
    uint32_t offset1;
    uint32_t offset2;
};

int32_t read_int_from_istream(std::istream& is) {
    int32_t val;
    is.read(reinterpret_cast<char*>(&val), sizeof(val));
    return val;

}

// Custom exception class for invalid index files
class InvalidIndexFile : public std::runtime_error {
    public:
        explicit InvalidIndexFile(const std::string& msg) : std::runtime_error(msg) {}
};

struct SyncmerParameters {
    const int k;
    const int s;
    const int t_syncmer;

    SyncmerParameters(int k, int s)
        : k(k)
          , s(s)
          , t_syncmer((k - s) / 2 + 1)
    {
        verify();
    }

    void verify() const {
        if (k <= 7 || k > 32) {
            //throw BadParameter("k not in [8,32]");
        }
        if (s > k) {
            //throw BadParameter("s is larger than k");
        }
        if ((k - s) % 2 != 0) {
            //throw BadParameter("(k - s) must be an even number to create canonical syncmers. Please set s to e.g. k-2, k-4, k-6, ...");
        }
    }

    bool operator==(const SyncmerParameters& other) const;
};

struct RandstrobeParameters {
    const int l;
    const int u;
    const uint64_t q;
    const int max_dist;
    const unsigned w_min;
    const unsigned w_max;

    RandstrobeParameters(int l, int u, uint64_t q, int max_dist, unsigned w_min, unsigned w_max)
        : l(l)
          , u(u)
          , q(q)
          , max_dist(max_dist)
          , w_min(w_min)
          , w_max(w_max)
    {
        verify();
    }

    bool operator==(const RandstrobeParameters& other) const;

    private:
    void verify() const {
        if (max_dist > 255) {
            //throw BadParameter("maximum seed length (-m <max_dist>) is larger than 255");
        }
        if (w_min > w_max) {
            //throw BadParameter("w_min is greater than w_max (choose different -l/-u parameters)");
        }
    }
};


/* Settings that influence index creation */
class IndexParameters {
    public:
        const size_t canonical_read_length;
        const SyncmerParameters syncmer;
        const RandstrobeParameters randstrobe;

        static const int DEFAULT = std::numeric_limits<int>::min();

        IndexParameters(size_t canonical_read_length, int k, int s, int l, int u, int q, int max_dist)
            : canonical_read_length(canonical_read_length)
              , syncmer(k, s)
              , randstrobe(l, u, q, max_dist, std::max(0, k / (k - s + 1) + l), k / (k - s + 1) + u)
    {
    }

        static IndexParameters read(std::istream& is) {
            size_t canonical_read_length = read_int_from_istream(is);
            int k = read_int_from_istream(is);
            int s = read_int_from_istream(is);
            int l = read_int_from_istream(is);
            int u = read_int_from_istream(is);
            int q = read_int_from_istream(is);
            int max_dist = read_int_from_istream(is);
            return IndexParameters(canonical_read_length, k, s, l, u, q, max_dist);
        }


        std::string filename_extension() const;
        void write(std::ostream& os) const;
        bool operator==(const IndexParameters& other) const;
        bool operator!=(const IndexParameters& other) const { return !(*this == other); }
};


// Helper function template: Read a vector from the input stream
template <typename T>
void read_vector(std::istream& is, std::vector<T>& v) {
    uint64_t size;
    v.clear();
    is.read(reinterpret_cast<char*>(&size), sizeof(size));
    v.resize(size);
    is.read(reinterpret_cast<char*>(v.data()), size * sizeof(T));
}

__global__ void gpu_find_single_thread(
        const RefRandstrobe* d_randstrobes,
        const bucket_index_t* d_randstrobe_start_indices,
        const randstrobe_hash_t* d_queries,
        size_t* d_positions,
        int num_queries,
        int bits
        ) {
    if (threadIdx.x == 0 && blockIdx.x == 0) { 
        for (int tid = 0; tid < num_queries; ++tid) {
            randstrobe_hash_t key = d_queries[tid];
            const unsigned int top_N = key >> (64 - bits);
            bucket_index_t position_start = d_randstrobe_start_indices[top_N];
            bucket_index_t position_end = d_randstrobe_start_indices[top_N + 1];

            if (position_start == position_end) {
                d_positions[tid] = static_cast<size_t>(-1); // No match
                continue;
            }

            bool found = false;
            for (bucket_index_t i = position_start; i < position_end; ++i) {
                if (d_randstrobes[i].hash == key) {
                    d_positions[tid] = i;
                    found = true;
                    break;
                }
            }

            if (!found) {
                d_positions[tid] = static_cast<size_t>(-1); // No match
            }
        }
    }
}



// GPU kernel for find
__global__ void gpu_find_bs(
        const RefRandstrobe* d_randstrobes,
        const bucket_index_t* d_randstrobe_start_indices,
        const randstrobe_hash_t* d_queries,
        size_t* d_positions,
        int num_queries,
        int bits
        ) {
    int toffset = blockIdx.x * blockDim.x + threadIdx.x;
    int sz = gridDim.x * blockDim.x;
    for(int id = 0, tid = toffset; id < 32; id++, tid += sz) {
        if (tid < num_queries) {
            randstrobe_hash_t key = d_queries[tid];
            const unsigned int top_N = key >> (64 - bits);
            bucket_index_t position_start = d_randstrobe_start_indices[top_N];
            bucket_index_t position_end = d_randstrobe_start_indices[top_N + 1];

            //d_positions[tid] = position_end - position_start;

            if (position_start == position_end) {
                d_positions[tid] = static_cast<size_t>(-1); // No match
                continue;
                //return;
            }

            bool has_find = 0;
            for (bucket_index_t i = position_start; i < position_end; ++i) {
                if (d_randstrobes[i].hash == key) {
                    d_positions[tid] = i;
                    has_find = 1;
                    break;
                    //return;
                }
            }

            if(has_find == 0) d_positions[tid] = static_cast<size_t>(-1); // No match
        }
    }
}


__global__ void gpu_find(
        const RefRandstrobe* d_randstrobes,
        const bucket_index_t* d_randstrobe_start_indices,
        const randstrobe_hash_t* d_queries,
        size_t* d_positions,
        int num_queries,
        int bits
        ) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_queries) {
        randstrobe_hash_t key = d_queries[tid];
        const unsigned int top_N = key >> (64 - bits);
        bucket_index_t position_start = d_randstrobe_start_indices[top_N];
        bucket_index_t position_end = d_randstrobe_start_indices[top_N + 1];

        //d_positions[tid] = position_end - position_start;

        if (position_start == position_end) {
            d_positions[tid] = static_cast<size_t>(-1); // No match
            return;
        }

        for (bucket_index_t i = position_start; i < position_end; ++i) {
            if (d_randstrobes[i].hash == key) {
                d_positions[tid] = i;
                return;
            }
        }

        d_positions[tid] = static_cast<size_t>(-1); // No match
    }
}

// Class representing the StrobemerIndex
class StrobemerIndex {
    public:
        std::vector<RefRandstrobe> randstrobes;  // Stores the randstrobes
        std::vector<bucket_index_t> randstrobe_start_indices;  // Start indices for hash buckets
        int bits;  // Number of bits for the hash table
        int filter_cutoff;  // Filter cutoff value

        // Read the index from a file
        void read(const std::string& filename) {
            errno = 0;
            std::ifstream ifs(filename, std::ios::binary);
            if (!ifs.is_open()) {
                throw InvalidIndexFile(filename + ": " + strerror(errno));
            }

            // Check magic number to verify file format
            union {
                char s[4];
                uint32_t v;
            } magic;
            ifs.read(magic.s, 4);
            if (magic.v != 0x01495453) {
                throw InvalidIndexFile("Index file has incorrect format (magic number mismatch)");
            }

            // Read file format version
            uint32_t file_format_version = read_int_from_istream(ifs);
            if (file_format_version != 2) {
                throw InvalidIndexFile("Unsupported index file format version");
            }

            // Skip over reserved chunk
            randstrobe_hash_t reserved_chunk_size;
            ifs.read(reinterpret_cast<char*>(&reserved_chunk_size), sizeof(reserved_chunk_size));
            ifs.seekg(reserved_chunk_size, std::ios_base::cur);

            // Read other parameters
            filter_cutoff = read_int_from_istream(ifs);
            bits = read_int_from_istream(ifs);
            const IndexParameters sti_parameters = IndexParameters::read(ifs);

            // Read randstrobes and start indices
            read_vector(ifs, randstrobes);
            read_vector(ifs, randstrobe_start_indices);
            if (randstrobe_start_indices.size() != (1u << bits) + 1) {
                throw InvalidIndexFile("randstrobe_start_indices vector is of the wrong size");
            }
        }

        // Find a key in the index
        size_t find(randstrobe_hash_t key) const {
            //constexpr int MAX_LINEAR_SEARCH = 4;
            const unsigned int top_N = key >> (64 - bits);
            bucket_index_t position_start = randstrobe_start_indices[top_N];
            bucket_index_t position_end = randstrobe_start_indices[top_N + 1];
            //return position_end - position_start;
            //std::cout << "top_N: " << top_N << ", range: " << position_start << " " << position_end << std::endl;

            if (position_start == position_end) {
                return static_cast<size_t>(-1); // No match
            }

            //if (position_end - position_start < MAX_LINEAR_SEARCH) {
            for (; position_start < position_end; ++position_start) {
                if (randstrobes[position_start].hash == key) return position_start;
                if (randstrobes[position_start].hash > key) break;
            }
            return static_cast<size_t>(-1); // No match
            //}

            //auto cmp = [](const RefRandstrobe& lhs, const randstrobe_hash_t rhs) { return lhs.hash < rhs; };
            //auto pos = std::lower_bound(randstrobes.begin() + position_start,
            //                            randstrobes.begin() + position_end,
            //                            key, cmp);

            //if (pos != randstrobes.end() && pos->hash == key) {
            //    return std::distance(randstrobes.begin(), pos);
            //}
            //return static_cast<size_t>(-1); // No match
        }


};


std::vector<uint64_t> readFileToVector(const std::string& filename) {
    std::vector<uint64_t> data;
    std::ifstream infile(filename);

    if (!infile.is_open()) {
        throw std::runtime_error("Failed to open file: " + filename);
    }

    std::string line;
    while (std::getline(infile, line)) {
        std::istringstream iss(line);
        uint64_t value;
        while (iss >> value) {
            data.push_back(value);
        }
    }

    infile.close();
    return data;
}


//#define use_managed

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <index_file_path>" << std::endl;
        return 1;
    }

    std::string index_file_path = argv[1];
    StrobemerIndex index;
    try {
        index.read(index_file_path);
    } catch (const std::exception& e) {
        std::cerr << "Error reading index file: " << e.what() << std::endl;
        return 1;
    }

	std::vector<randstrobe_hash_t> queries = readFileToVector("seed_info.txt");
    int num_queries = queries.size();
    printf("size %d, data[0] %lu\n", num_queries, queries[0]);
    double t0;

    size_t* d_positions;
    randstrobe_hash_t* d_queries;
    RefRandstrobe* d_randstrobes;
    bucket_index_t* d_randstrobe_start_indices;

    size_t* h_positions;
    size_t* positions = (size_t *)malloc(num_queries * sizeof(size_t));
    memset(positions, 0, num_queries * sizeof(size_t));
    randstrobe_hash_t* h_queries;

    hipHostAlloc(&h_positions, num_queries * sizeof(size_t), hipHostMallocDefault);
    hipHostAlloc(&h_queries, num_queries * sizeof(randstrobe_hash_t), hipHostMallocDefault);
    memcpy(h_queries, &(queries[0]), num_queries * sizeof(randstrobe_hash_t));

    t0 = GetTime();
    hipMalloc(&d_positions, num_queries * sizeof(size_t));
    hipMalloc(&d_queries, num_queries * sizeof(randstrobe_hash_t));
    hipMalloc(&d_randstrobes, index.randstrobes.size() * sizeof(RefRandstrobe));
    hipMalloc(&d_randstrobe_start_indices, index.randstrobe_start_indices.size() * sizeof(bucket_index_t));
    hipMemset(d_positions, 0, num_queries * sizeof(size_t));
    hipMemset(d_queries, 0, num_queries * sizeof(randstrobe_hash_t));
    hipMemset(d_randstrobes, 0, index.randstrobes.size() * sizeof(RefRandstrobe));
    hipMemset(d_randstrobe_start_indices, 0, index.randstrobe_start_indices.size() * sizeof(bucket_index_t));
    std::cout << "malloc execution time: " << GetTime() - t0 << " seconds, size " << num_queries * sizeof(size_t) + num_queries * sizeof(randstrobe_hash_t) + index.randstrobes.size() * sizeof(RefRandstrobe) + index.randstrobe_start_indices.size() * sizeof(bucket_index_t) << std::endl;

    t0 = GetTime();
    hipMemcpy(d_randstrobes, index.randstrobes.data(), index.randstrobes.size() * sizeof(RefRandstrobe), hipMemcpyHostToDevice);
    hipMemcpy(d_randstrobe_start_indices, index.randstrobe_start_indices.data(), index.randstrobe_start_indices.size() * sizeof(bucket_index_t), hipMemcpyHostToDevice);
    std::cout << "memcpy1 execution time: " << GetTime() - t0 << " seconds, size " << index.randstrobes.size() * sizeof(RefRandstrobe) + index.randstrobe_start_indices.size() * sizeof(bucket_index_t) << std::endl;

    std::thread gpu_thread([&]() {
        sleep(5);
        for(int i = 0; i < 8; i++) {
            t0 = GetTime();
            hipMemcpy(d_queries, h_queries, num_queries * sizeof(randstrobe_hash_t), hipMemcpyHostToDevice);
            std::cout << "memcpy2 execution time: " << GetTime() - t0 << " seconds, size " << num_queries * sizeof(randstrobe_hash_t) << std::endl;

            t0 = GetTime();
            int threads_per_block = 1 << (6 + i % 4);
            int blocks_per_grid = (num_queries + threads_per_block - 1) / threads_per_block;
            gpu_find<<<blocks_per_grid, threads_per_block>>>(d_randstrobes, d_randstrobe_start_indices, d_queries, d_positions, num_queries, index.bits);
            //gpu_find_bs<<<blocks_per_grid / 32, threads_per_block>>>(d_randstrobes, d_randstrobe_start_indices, d_queries, d_positions, num_queries, index.bits);
            hipDeviceSynchronize(); // Ensure all threads are finished
            std::cout << "synchronize execution time: " << GetTime() - t0 << " seconds" << std::endl;

            t0 = GetTime();
            hipMemcpy(h_positions, d_positions, num_queries * sizeof(size_t), hipMemcpyDeviceToHost);
            std::cout << "memcpy back execution time: " << GetTime() - t0 << " seconds" << std::endl;
        }
    });

    t0 = GetTime();
#pragma omp parallel for num_threads(16)
    for (int i = 0; i < num_queries; ++i) {
        positions[i] = index.find(queries[i]);
    }
    std::cout << "CPU execution time: " << GetTime() - t0 << " seconds" << std::endl;



    size_t check_sum = 0;
    for (size_t i = 0; i < 10; ++i) {
        int id = rand() % num_queries;
        std::cout << "Query " << id << ": Position " << positions[id] << std::endl;
        check_sum += positions[id];
    }
    std::cout << "check sum is " << check_sum << std::endl;

    gpu_thread.join();
    t0 = GetTime();
    hipFree(d_randstrobes);
    hipFree(d_randstrobe_start_indices);
    hipFree(d_queries);
    hipFree(d_positions);
    std::cout << "free execution time: " << GetTime() - t0 << " seconds" << std::endl;

    return 0;
}
